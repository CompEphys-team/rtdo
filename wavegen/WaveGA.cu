/*--------------------------------------------------------------------------
Author: Daniel Saska

Institute: Informatics
University of Sussex
Brighton BN1 9QJ, UK

email to:  ds376@sussex.ac.uk

initial version: 2014-09-09

--------------------------------------------------------------------------*/

//--------------------------------------------------------------------------
/*! \file VClampGA.cu

\brief Main entry point for the GeNN project demonstrating realtime fitting of a neuron with a GA running mostly on the GPU.
*/
//--------------------------------------------------------------------------


// minimum duration of voltage step
#define MINSTEP 2.0  
// minimal voltage
#define MINV -100.0
// maximal voltage
#define MAXV 50.0
// minimal observation time window
#define MINT 5.0
// maximal observation time window
#define MAXT 100.0
// observation time (fixed)
#define OT 100.0
// total simulation time
#define SIM_TIME 200.0
// initial dispersion of V steps
#define VSTEPINI 40.0
// initial "baseline" for V steps
#define VSTEP0 -60.0
// initial maximum width of steps (ms)
#define STEPWDINI 100.0
// minimal position of first step
#define MINSTEPT 10.0
// number of GA NGENs
int NGEN = 500;
// factor of reducing mutateA
#define MUTATEFAC 0.995
// number of steps in the voltage signals
#define NVSTEPS 3

#define TOTALT 200

// the size of random mutations
double mutateA = 10.0;


#include "WaveGA.h"
#include "waveHelper.h"
#include <hip/hip_runtime.h>

//--------------------------------------------------------------------------
/*! \brief This function is the entry point for running the project
*/
//--------------------------------------------------------------------------


extern "C" inputSpec wavegen(int focusParam, int nGenerations, bool *stopFlag)
{
    double gaBalance = 1.0 / (NPARAM - 1.0);
    NGEN = nGenerations;

	//-----------------------------------------------------------------
    // Initialize population
	vector<inputSpec> stims;
	wave_pop_init( stims, GAPOP );
	size_t * sn = new size_t[NPOP];

	//-----------------------------------------------------------------
	// build the neuronal circuitery

	NNmodel model;
	modelDefinition( model );
	allocateMem();
	initialize();
    rtdo_init_bridge();
	
    //------------------------------------------------------
    // Get steady-state variable values at holding potential
    stepVGHH[0] = VSTEP0;
    for ( double t = 0.0; t < 10000.0; t += DT ) {
        calcSingleNeuronCPU(t);
    }
    scalar holdingVar[NVAR];
    for ( int i = 0; i < NVAR; i++ )
        holdingVar[i] = mvar[i][0];


	unsigned int VSize = NPOP*theSize( model.ftype );

    for (size_t generation = 0; generation < NGEN && !*stopFlag; ++generation)
	{
        for ( int i = 0; i < NVAR; i++ ) {
            for ( int j = 0; j < NPOP; j++ ) {
                mvar[i][j] = holdingVar[i];
            }
        }
        for ( int j = 0; j < NPOP; j++ ) {
            errHH[j] = 0.0;
        }

        var_init_auto_detune();
        memset( sn, 0x00000000, NPOP * sizeof( size_t ) );
        otHH = OT;

		for (size_t i = 0; i < stims.size(); ++i)
		{
            for (size_t j = 0; j < NPARAM + 1; j++)
			{
                float tmp = (stims[i].ot + stims[i].dur);
                CHECK_CUDA_ERRORS( hipMemcpy( &d_oteHH[i * (NPARAM + 1) + j], &tmp, sizeof( float ), hipMemcpyHostToDevice ) );
			}
		}
		for (double t = 0.0; t < SIM_TIME; t += DT)
        {
            stepTimeGPU( t );

			for (size_t i = 0; i < stims.size(); ++i)
			{
				if ((sn[i] < stims[i].N) && ((t - DT < stims[i].st[sn[i]]) && (t >= stims[i].st[sn[i]]) || (stims[i].st[sn[i]] == 0))) 
				{
                    for (size_t j = 0; j < NPARAM + 1; ++j)
					{
						float tmp = stims[i].V[sn[i]];
                        CHECK_CUDA_ERRORS( hipMemcpy( &d_stepVGHH[i * (NPARAM + 1) + j], &tmp, sizeof( float ), hipMemcpyHostToDevice ) );
					}
					++sn[i];
				}
			}

		}

        CHECK_CUDA_ERRORS( hipMemcpy( errHH, d_errHH, VSize, hipMemcpyDeviceToHost ) );
        for (size_t i = 0; i < stims.size(); ++i)
        {
            stims[i].fit = 0.0;
            for (size_t j = 1; j < NPARAM + 1; j++)
            {
                if (j == focusParam+1)
                {
                    stims[i].fit += errHH[i * (NPARAM + 1) + j];
                }
                else
                {
                    stims[i].fit -= errHH[i * (NPARAM + 1) + j] * gaBalance;
                }
            }
        }
        procreatePop( stims );
        cout << "Generation " << generation << "'s best stimulus:" << endl;
        cout << stims[0] << endl;
	}
	delete[] sn;
    return stims[0];
}
