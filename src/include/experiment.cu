#ifndef EXPERIMENT_CU
#define EXPERIMENT_CU

#include "experimentlibrary.h"
#include "cuda_helper.h" // For syntax highlighting only

void libInit(ExperimentLibrary::Pointers &pointers, size_t numModels)
{
    pointers.pushErr = [&pointers, numModels](){
        CHECK_CUDA_ERRORS(hipMemcpy(pointers.d_err, pointers.err, numModels * sizeof(scalar), hipMemcpyHostToDevice))
    };
    pointers.pullErr = [&pointers, numModels](){
        CHECK_CUDA_ERRORS(hipMemcpy(pointers.err, pointers.d_err, numModels * sizeof(scalar), hipMemcpyDeviceToHost))
    };

    allocateMem();
    initialize();
}

extern "C" void libExit(ExperimentLibrary::Pointers &pointers)
{
    freeMem();
    pointers.pushErr = pointers.pullErr = nullptr;
}

extern "C" void resetDevice()
{
    hipDeviceReset();
}

#endif
