#ifndef WAVEGEN_CU
#define WAVEGEN_CU

#include "wavegen_globals.h"
#include "cuda_helper.h" // For syntax highlighting only

__device__ WaveStats *dd_wavestats;
__device__ Stimulation *dd_waveforms;

void allocateGroupMem()
{
    using namespace Wavegen_Global;
    hipHostAlloc(&wavestats, MM_NumGroups * sizeof(WaveStats), hipHostMallocPortable);
        deviceMemAllocate(&d_wavestats, dd_wavestats, MM_NumGroups * sizeof(WaveStats));
    hipHostAlloc(&waveforms, MM_NumGroups * sizeof(Stimulation), hipHostMallocPortable);
        deviceMemAllocate(&d_waveforms, dd_waveforms, MM_NumGroups * sizeof(Stimulation));

    hipHostAlloc(&clear_wavestats, MM_NumGroups * sizeof(WaveStats), hipHostMallocPortable);
    for ( unsigned i = 0; i < MM_NumGroups; i++ )
        clear_wavestats[i] = {};
}
void clearStats()
{
    using namespace Wavegen_Global;
    CHECK_CUDA_ERRORS(hipMemcpy(d_wavestats, clear_wavestats, MM_NumGroups * sizeof(WaveStats), hipMemcpyHostToDevice))
}
void pullStats()
{
    using namespace Wavegen_Global;
    CHECK_CUDA_ERRORS(hipMemcpy(wavestats, d_wavestats, MM_NumGroups * sizeof(WaveStats), hipMemcpyDeviceToHost))
}
void pushWaveforms()
{
    using namespace Wavegen_Global;
    CHECK_CUDA_ERRORS(hipMemcpy(d_waveforms, waveforms, MM_NumGroups * sizeof(Stimulation), hipMemcpyHostToDevice))
}
void pullWaveforms()
{
    using namespace Wavegen_Global;
    CHECK_CUDA_ERRORS(hipMemcpy(waveforms, d_waveforms, MM_NumGroups * sizeof(Stimulation), hipMemcpyDeviceToHost))
}
void freeGroupMem()
{
    using namespace Wavegen_Global;
    hipHostFree(wavestats);
    hipHostFree(clear_wavestats);
    CHECK_CUDA_ERRORS(hipFree(d_wavestats));
    hipHostFree(waveforms);
    CHECK_CUDA_ERRORS(hipFree(d_waveforms));
}

void libManualInit(MetaModel &m) // Must be called separately (through Wavegen_Global::init())
{
    allocateMem();
    allocateGroupMem();
    initialize();
    clearStats();
    Wavegen_Global::populate(m);
}

void __attribute__ ((constructor)) libInit()
{
    Wavegen_Global::init =& libManualInit;
    Wavegen_Global::push =& pushHHStateToDevice;
    Wavegen_Global::pull =& pullHHStateFromDevice;
    Wavegen_Global::step =& stepTimeGPU;
    Wavegen_Global::reset =& initialize;
    Wavegen_Global::pullStats =& pullStats;
    Wavegen_Global::clearStats =& clearStats;
    Wavegen_Global::pushWaveforms =& pushWaveforms;
    Wavegen_Global::pullWaveforms =& pullWaveforms;

    Wavegen_Global::t =& t;
    Wavegen_Global::iT =& iT;
}

void libExit()
{
    freeMem();
    freeGroupMem();
    hipDeviceReset();
    Wavegen_Global::init = 0;
    Wavegen_Global::push = 0;
    Wavegen_Global::pull = 0;
    Wavegen_Global::step = 0;
    Wavegen_Global::reset = 0;
    Wavegen_Global::pullStats = 0;
    Wavegen_Global::clearStats = 0;
    Wavegen_Global::pushWaveforms = 0;
    Wavegen_Global::pullWaveforms = 0;

    Wavegen_Global::t = 0;
    Wavegen_Global::iT = 0;
}

#endif
