#ifndef WAVEGEN_CU
#define WAVEGEN_CU

#include "wavegenconstructor.h"
#include "cuda_helper.h" // For syntax highlighting only

static __device__ WaveStats *dd_wavestats;
static __device__ Stimulation *dd_waveforms;

void allocateGroupMem(WavegenConstructor::Pointers &pointers)
{
    hipHostAlloc(&pointers.wavestats, MM_NumGroups * sizeof(WaveStats), hipHostMallocPortable);
        deviceMemAllocate(&pointers.d_wavestats, dd_wavestats, MM_NumGroups * sizeof(WaveStats));
    hipHostAlloc(&pointers.waveforms, MM_NumGroups * sizeof(Stimulation), hipHostMallocPortable);
        deviceMemAllocate(&pointers.d_waveforms, dd_waveforms, MM_NumGroups * sizeof(Stimulation));

    hipHostAlloc(&pointers.clear_wavestats, MM_NumGroups * sizeof(WaveStats), hipHostMallocPortable);
    for ( unsigned i = 0; i < MM_NumGroups; i++ )
        pointers.clear_wavestats[i] = {};
}

void freeGroupMem(WavegenConstructor::Pointers &pointers)
{
    hipHostFree(pointers.wavestats);
    hipHostFree(pointers.clear_wavestats);
    CHECK_CUDA_ERRORS(hipFree(pointers.d_wavestats));
    hipHostFree(pointers.waveforms);
    CHECK_CUDA_ERRORS(hipFree(pointers.d_waveforms));
}

void libInit(WavegenConstructor::Pointers &pointers, size_t numGroups, size_t numModels)
{
    pointers.clearStats = [&pointers, numGroups](){
        CHECK_CUDA_ERRORS(hipMemcpy(pointers.d_wavestats, pointers.clear_wavestats, numGroups * sizeof(WaveStats), hipMemcpyHostToDevice))
    };
    pointers.pullStats = [&pointers, numGroups](){
        CHECK_CUDA_ERRORS(hipMemcpy(pointers.wavestats, pointers.d_wavestats, numGroups * sizeof(WaveStats), hipMemcpyDeviceToHost))
    };
    pointers.pushWaveforms = [&pointers, numGroups](){
        CHECK_CUDA_ERRORS(hipMemcpy(pointers.d_waveforms, pointers.waveforms, numGroups * sizeof(Stimulation), hipMemcpyHostToDevice))
    };
    pointers.pullWaveforms = [&pointers, numGroups](){
        CHECK_CUDA_ERRORS(hipMemcpy(pointers.waveforms, pointers.d_waveforms, numGroups * sizeof(Stimulation), hipMemcpyDeviceToHost))
    };
    pointers.pushErr = [&pointers, numModels](){
        CHECK_CUDA_ERRORS(hipMemcpy(pointers.d_err, pointers.err, numModels * sizeof(scalar), hipMemcpyHostToDevice))
    };
    pointers.pullErr = [&pointers, numModels](){
        CHECK_CUDA_ERRORS(hipMemcpy(pointers.err, pointers.d_err, numModels * sizeof(scalar), hipMemcpyDeviceToHost))
    };

    allocateMem();
    allocateGroupMem(pointers);
    initialize();
    pointers.clearStats();
}

extern "C" void libExit(WavegenConstructor::Pointers &pointers)
{
    freeMem();
    freeGroupMem(pointers);
    pointers.clearStats = pointers.pullStats = pointers.pushWaveforms = pointers.pullWaveforms = pointers.pushErr = pointers.pullErr = nullptr;
}

extern "C" void resetDevice()
{
    hipDeviceReset();
}

#endif
