#include "hip/hip_runtime.h"
#ifndef UNIVERSAL_CU
#define UNIVERSAL_CU

#include "universallibrary.h"
#include "cuda_helper.h"

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/count.h>

static scalar *target = nullptr, *d_target = nullptr;
static __constant__ scalar *dd_target = nullptr;
static unsigned int target_size = 0, latest_target_size = 0;

static scalar *timeseries = nullptr, *d_timeseries = nullptr;
static __constant__ scalar *dd_timeseries = nullptr;
static unsigned int timeseries_size = 0, latest_timeseries_size = 0;

static __constant__ iStimulation singular_stim;
static __constant__ iObservations singular_obs;

static __constant__ scalar singular_clampGain;
static __constant__ scalar singular_accessResistance;
static __constant__ int singular_iSettleDuration;
static __constant__ scalar singular_Imax;
static __constant__ scalar singular_dt;

static __constant__ size_t singular_targetOffset;

// profiler memory space
static constexpr unsigned int NPAIRS = NMODELS/2;
static scalar *d_gradient;
static constexpr unsigned int gradientSz = NPAIRS * (NPAIRS - 1); // No diagonal

// elementary effects wg / clustering memory space
static scalar *clusters = nullptr, *d_clusters = nullptr;
static unsigned int clusters_size = 0;

static int *clusterLen = nullptr, *d_clusterLen = nullptr;
static unsigned int clusterLen_size = 0;

static unsigned int *d_clusterMasks = nullptr;
static unsigned int clusterMasks_size = 0;

static scalar *clusterCurrent = nullptr, *d_clusterCurrent = nullptr;
static unsigned int clusterCurrent_size = 0;

static scalar *sections = nullptr, *d_sections = nullptr;
static unsigned int sections_size = 0;

static scalar *d_currents= nullptr;
static unsigned int currents_size = 0;

static iObservations *clusterObs = nullptr, *d_clusterObs = nullptr;
static unsigned int clusterObs_size = 0;

static __constant__ scalar deltabar[NPARAMS];

void libInit(UniversalLibrary &lib, UniversalLibrary::Pointers &pointers)
{
    pointers.pushV = [](void *hostptr, void *devptr, size_t size){
        CHECK_CUDA_ERRORS(hipMemcpy(devptr, hostptr, size, hipMemcpyHostToDevice))
    };
    pointers.pullV = [](void *hostptr, void *devptr, size_t size){
        CHECK_CUDA_ERRORS(hipMemcpy(hostptr, devptr, size, hipMemcpyDeviceToHost));
    };

    pointers.target =& target;
    pointers.output =& timeseries;

    pointers.clusters =& clusters;
    pointers.clusterCurrent =& clusterCurrent;
    pointers.clusterPrimitives =& sections;
    pointers.clusterObs =& clusterObs;

    allocateMem();
    initialize();

    hipGetSymbolAddress((void **)&lib.stim.singular_v, singular_stim);
    hipGetSymbolAddress((void **)&lib.obs.singular_v, singular_obs);

    hipGetSymbolAddress((void **)&lib.clampGain.singular_v, singular_clampGain);
    hipGetSymbolAddress((void **)&lib.accessResistance.singular_v, singular_accessResistance);
    hipGetSymbolAddress((void **)&lib.iSettleDuration.singular_v, singular_iSettleDuration);
    hipGetSymbolAddress((void **)&lib.Imax.singular_v, singular_Imax);
    hipGetSymbolAddress((void **)&lib.dt.singular_v, singular_dt);

    hipGetSymbolAddress((void **)&lib.targetOffset.singular_v, singular_targetOffset);

    CHECK_CUDA_ERRORS(hipMalloc(&d_gradient, gradientSz * sizeof(scalar)));
}

extern "C" void libExit(UniversalLibrary::Pointers &pointers)
{
    freeMem();
    pointers.pushV = pointers.pullV = nullptr;
}

extern "C" void resetDevice()
{
    hipDeviceReset();
}

extern "C" void resizeTarget(size_t newSize)
{
    resizeArrayPair(target, d_target, target_size, newSize);
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dd_target), &d_target, sizeof(scalar*)));
    latest_target_size = newSize;
}

extern "C" void pushTarget()
{
    CHECK_CUDA_ERRORS(hipMemcpy(d_target, target, latest_target_size * sizeof(scalar), hipMemcpyHostToDevice))
}

extern "C" void resizeOutput(size_t newSize)
{
    resizeArrayPair(timeseries, d_timeseries, timeseries_size, newSize);
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dd_timeseries), &d_timeseries, sizeof(scalar*)));
    latest_timeseries_size = newSize;
}

extern "C" void pullOutput()
{
    CHECK_CUDA_ERRORS(hipMemcpy(timeseries, d_timeseries, latest_timeseries_size * sizeof(scalar), hipMemcpyDeviceToHost))
}


// Code adapated from Justin Luitjens, <https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/>
// Note, shuffle is only supported on compute capability 3.x and higher
__device__ inline scalar warpReduceSum(scalar val, int cutoff = warpSize)
{
    for ( int offset = 1; offset < cutoff; offset *= 2 )
        val += __shfl_xor_sync(0xffffffff, val, offset);
    return val;
}



/// ******************************************************************************************************************************
///  >============================     Profiler kernel & host function      ====================================================<
/// ******************************************************************************************************************************

// Compute the current deviation of both tuned and untuned models against each tuned model
// Models are interleaved (even id = tuned, odd id = detuned) in SamplingProfiler
__global__ void compute_gradient(int nSamples, int stride, scalar *targetParam, scalar *gradient)
{
    unsigned int xThread = blockIdx.x * blockDim.x + threadIdx.x; // probe
    unsigned int yThread = blockIdx.y * blockDim.y + threadIdx.y; // reference
    unsigned int x,y;
    if ( xThread < yThread ) { // transpose subdiagonal half of the top-left quadrant to run on the supradiagonal half of bottom-right quadrant
        // the coordinate transformation is equivalent to squashing the bottom-right supradiagonal triangle to the left border,
        // then flipping it up across the midline.
        x = xThread + NPAIRS - yThread; // xnew = x + n-y
        y = NPAIRS - yThread - 1;       // ynew = n-y - 1
    } else {
        x = xThread;
        y = yThread;
    }

    scalar err_tx_ty = 0., err_tx_dy = 0., err_dx_ty = 0., err;
    int i = 0;
    for ( ; i < nSamples; i += stride ) {
        scalar xval = dd_timeseries[2*x + NMODELS*i];
        scalar yval = dd_timeseries[2*y + NMODELS*i];

        err = xval - yval;
        err_tx_ty += err*err;

        err = xval - dd_timeseries[2*y+1 + NMODELS*i];
        err_tx_dy += err*err;

        err = yval - dd_timeseries[2*x+1 + NMODELS*i];
        err_dx_ty += err*err;
    }

    i = nSamples/stride; // Using i as nSamplesUsed
    err_tx_ty = std::sqrt(err_tx_ty / i);
    err_tx_dy = std::sqrt(err_tx_dy / i);
    err_dx_ty = std::sqrt(err_dx_ty / i);

    if ( x != y ) { // Ignore diagonal (don't probe against self)
        // invert sign as appropriate, such that detuning in the direction of the reference is reported as positive
        i = (1 - 2 * (targetParam[2*x] < targetParam[2*y])); // using i as sign

        // fractional change in error ( (d_err-t_err)/t_err) "how much does the error improve by detuning, relative to total error?")
        err = ((err_dx_ty / err_tx_ty) - 1) * i;

        // Put invalid values to the end of the scale, positive or negative; heuristically balance both sides
        if ( ::isnan(err) )
            err = i * SCALAR_MAX;

        // Addressing: Squish the diagonal out to prevent extra zeroes
        gradient[xThread + NPAIRS*yThread - yThread - (xThread>yThread)] = err;

        err = (1 - (err_tx_dy / err_tx_ty)) * i; // = ((err_tx_dy / err_tx_ty) - 1) * -i
        if ( ::isnan(err) )
            err = -i * SCALAR_MAX;
        gradient[xThread + NPAIRS*yThread - yThread - (xThread>yThread) + (NPAIRS-1)*(NPAIRS/2)] = err;
    }
}

struct is_positive : public thrust::unary_function<scalar, bool>
{
    __host__ __device__ bool operator()(scalar x){
        return x > 0;
    }
};

extern "C" void profile(int nSamples, int stride, scalar *d_targetParam, double &accuracy, double &median_norm_gradient)
{
    dim3 block(32, 16);
    dim3 grid(NPAIRS/32, NPAIRS/32);
    compute_gradient<<<grid, block>>>(nSamples, stride, d_targetParam, d_gradient);

    thrust::device_ptr<scalar> gradient = thrust::device_pointer_cast(d_gradient);
    thrust::sort(gradient, gradient + gradientSz);

    double nPositive = thrust::count_if(gradient, gradient + gradientSz, is_positive());
    accuracy = nPositive / gradientSz;

    scalar median_g[2];
    CHECK_CUDA_ERRORS(hipMemcpy(median_g, d_gradient + gradientSz/2, 2*sizeof(scalar), hipMemcpyDeviceToHost));
    median_norm_gradient = (median_g[0] + median_g[1]) / 2;
}





/// ******************************************************************************************************************************
///  >============================     Elementary Effects WG: Clustering    ====================================================<
/// ******************************************************************************************************************************
static constexpr int STIMS_PER_CLUSTER_BLOCK = 16;
static constexpr int PARTITION_SIZE = 32;

/**
 * @brief build_section_primitives chops the EE traces in d_timeseries into deltabar-normalised deviation vectors ("sections")
 *          representing up to secLen ticks. Sections are chunked into partitions of PARTITION_SIZE=32 sections each.
 *          Deviation vectors represent the mean deviation per tick, normalised to deltabar, caused by a single detuning.
 *          Note, this kernel expects the EE traces to be generated using TIMESERIES_COMPARE_NONE
 * @param out_sections is the output, laid out as [stimIdx][paramIdx][partitionIdx][secIdx (local to partition)].
 * @param out_current is the mean current within each section, laid out as [stimIdx][partitionIdx][secIdx].
 */
__global__ void build_section_primitives(const int trajLen,
                                         const int nTraj,
                                         const int nStims,
                                         const int duration,
                                         const int secLen,
                                         const int nPartitions,
                                         scalar *out_sections,
                                         scalar *out_current)
{
    const int warpid = threadIdx.x / 32; // acts as block-local stim idx
    const int laneid = threadIdx.x & 31;
    const int stimIdx = (blockIdx.x * blockDim.x + threadIdx.x) / 32; // global stim idx; one stim per warp
    const int nTraces = trajLen*nTraj; // Total number of traces per stim, including starting point models
    const int nUsefulTraces = (trajLen-1)*nTraj; // Number of mid-trajectory models per stim
    const int paramIdx_after_end_of_final_traj = nUsefulTraces % NPARAMS; // First param idx with one contrib fewer than the preceding ones
    const int lane0_offset = stimIdx * nTraces;
    const int nLoads = (nTraces + 31) & 0xffffffe0;
    const iObservations obs = dd_obsUNI[stimIdx*nTraces];
    int nextObs = 0;

    volatile __shared__ scalar sh_contrib[STIMS_PER_CLUSTER_BLOCK][NPARAMS][PARTITION_SIZE + 1];
    volatile __shared__ scalar sh_current[STIMS_PER_CLUSTER_BLOCK][PARTITION_SIZE + 1];

    for ( int i = threadIdx.x; i < STIMS_PER_CLUSTER_BLOCK*NPARAMS*(PARTITION_SIZE+1); i += blockDim.x )
        *((scalar*)sh_contrib + i) = 0;
    for ( int i = threadIdx.x; i < STIMS_PER_CLUSTER_BLOCK*(PARTITION_SIZE+1); i += blockDim.x )
        *((scalar*)sh_current + i) = 0;
    __syncthreads();

    if ( stimIdx >= nStims )
        return;

    unsigned int secIdx = 0;
    int trueSecLen_static;
    int t = 0;
    while ( t < duration ) {
        int trueSecLen = 0;
        __syncwarp();
        for ( int tEnd = t + secLen; t < tEnd; t++ ) { // Note, t<duration guaranteed by obs.stop
            if ( nextObs < iObservations::maxObs && t >= obs.start[nextObs] ) {
                if ( t < obs.stop[nextObs] ) {
                    for ( int i = laneid; i < nLoads; i += warpSize ) {
                        const int paramIdx = (i - 1 - (i/trajLen)) % NPARAMS;
                        scalar current_mylane = dd_timeseries[t*NMODELS + lane0_offset + i];
                        scalar current_prevlane = __shfl_up_sync(0xffffffff, current_mylane, 1);
                        scalar diff = scalarfabs(current_prevlane - current_mylane);
                        if ( i < nTraces ) {
                            if ( i % trajLen != 0 )
                                atomicAdd((scalar*)&sh_contrib[warpid][paramIdx][secIdx&31], diff);
                            current_mylane = scalarfabs(current_mylane);
                        } else {
                            current_mylane = 0;
                        }
                        current_mylane = warpReduceSum(current_mylane);
                        if ( laneid == 0 )
                            sh_current[warpid][secIdx&31] += current_mylane;
                    }
                    ++trueSecLen;
                } else {
                    ++nextObs;
                }
            }
        }
        if ( laneid == (secIdx&31) )
            trueSecLen_static = trueSecLen;

        if ( ((++secIdx) & 31) == 0 || t >= duration ) {
            __syncwarp();
            const int partitionIdx = (secIdx-1) >> 5;
            int nContrib = nUsefulTraces/NPARAMS + 1;
            if ( t < duration || laneid <= (secIdx&31) ) {
                for ( int paramIdx = 0; paramIdx < NPARAMS; paramIdx++ ) {
                    if ( paramIdx == paramIdx_after_end_of_final_traj )
                        --nContrib;
                    out_sections[stimIdx * NPARAMS * nPartitions * PARTITION_SIZE
                            + paramIdx * nPartitions * PARTITION_SIZE
                            + partitionIdx * PARTITION_SIZE
                            + laneid]
                            = trueSecLen_static
                              ? sh_contrib[warpid][paramIdx][laneid] / (trueSecLen_static * deltabar[paramIdx] * nContrib)
                              : 0;
                    sh_contrib[warpid][paramIdx][laneid] = 0;
                }
            }

            out_current[stimIdx * nPartitions * PARTITION_SIZE
                    + partitionIdx * PARTITION_SIZE
                    + laneid]
                    = sh_current[warpid][laneid] / (trueSecLen_static * nTraces);
            sh_current[warpid][laneid] = 0;
        }
    }
}

/**
 * @brief compare_within_partition compares all sections in a partition to each other, recording a similarity for each
 * @param myContrib is a section's deviation vector
 * @param dotp_threshold
 * @return a bitmask flagging each above-threshold similar section
 */
__device__ unsigned int compare_within_partition(const Parameters myContrib,
                                                 const scalar norm,
                                                 const scalar dotp_threshold)
{
    const unsigned laneid = threadIdx.x & 31;
    unsigned int mask = 1<<laneid;
    Parameters target_contrib;

    for ( int offset = 1; offset < 17; offset++ ) {
        int target = (laneid + offset)&31;

        // Compare against target
        target_contrib.shfl(myContrib, target);
        scalar target_norm = __shfl_sync(0xffffffff, norm, target);
        scalar dotp = myContrib.dotp(target_contrib);
        if ( dotp > 0 )
            dotp /= (norm * target_norm);

        // Process my own work
        if ( dotp > dotp_threshold ) {
            mask |= 1 << target;
        }

        // Retrieve the work of the thread that targeted me, and process that, too
        target = (laneid + 32 - offset)&31;
        dotp = __shfl_sync(0xffffffff, dotp, target);
        if ( offset < 16 && dotp > dotp_threshold ) {
            mask |= 1 << target;
        }
    }
    return mask;
}

/**
 * @brief compare_within_partition compares all sections in a partition to each other, recording a similarity for each
 * @param myContrib is a section's deviation vector
 * @param dotp_threshold
 * @return a bitmask flagging each above-threshold similar section
 */
__device__ unsigned int compare_between_partitions(const Parameters reference,
                                                   Parameters target,
                                                   const scalar ref_norm,
                                                   const scalar dotp_threshold,
                                                   unsigned int &target_mask)
{
    const unsigned laneid = threadIdx.x & 31;
    unsigned int ref_mask = 0;
    target_mask = 0;
    scalar target_norm = std::sqrt(target.dotp(target));
    const int srcLane = (laneid+1) & 31;
    for ( int i = 0; i < warpSize; i++ ) {
        // Compare against target
        scalar dotp = reference.dotp(target);
        if ( dotp > 0 )
            dotp /= (ref_norm * target_norm);
        if ( dotp > dotp_threshold ) {
            // Update reference
            ref_mask |= 1 << ((laneid+i)&31);

            // Update target
            target_mask |= 1 << laneid;
        }

        // Shuffle targets down (except after the final comparison)
        if ( i < 31 ) {
            target.shfl(target, srcLane);
            target_norm = __shfl_sync(0xffffffff, target_norm, srcLane);
        }
        // shuffle target mask down 32 times to return it to its original lane
        target_mask = __shfl_sync(0xffffffff, target_mask, srcLane);
    }
    return ref_mask;
}

template <typename T>
__device__ unsigned int warpReduceMaxIdx(unsigned int idx, T value)
{
    for ( unsigned int i = 1; i < warpSize; i *= 2 ) {
        T cmp_value = __shfl_down_sync(0xffffffff, value, i);
        unsigned int cmp_idx = __shfl_down_sync(0xffffffff, idx, i);
        if ( cmp_value > value ) {
            value = cmp_value;
            idx = cmp_idx;
        }
    }
    return __shfl_sync(0xffffffff, idx, 0);
}

/**
 * @brief exactClustering is a non-heuristic clustering implementation. It takes the outputs from build_section_primitives, extracting
 * for each stim a set of clusters with associated iObservations, Euclidean normal deviation vector, and mean current.
 */
__global__ void exactClustering(const int nPartitions,
                                const scalar dotp_threshold,
                                const int secLen,
                                const int minClusterLen,
                                scalar *in_contrib, /* [stimIdx][paramIdx][secIdx] */
                                scalar *in_current, /* [stimIdx][secIdx] */
                                scalar *out_clusters, /* [stimIdx][clusterIdx][paramIdx] */
                                scalar *out_clusterCurrent, /* [stimIdx][clusterIdx] */
                                iObservations *out_observations, /* [stimIdx][clusterIdx] */
                                unsigned int *out_masks, /* [stimIdx][partitionIdx][secIdx], intermediate only */
                                const unsigned int shmem_size /* in uints. Minimum nSecs+32, preferably much more for obs timestamp leeway */
                                )
{
    const unsigned laneid = threadIdx.x & 31;
    const unsigned warpid = threadIdx.x >> 5;
    const unsigned int nSecs = 32 * nPartitions;

    extern __shared__ unsigned int shmem[];
    for ( int i = threadIdx.x; i < shmem_size; i += blockDim.x )
        shmem[i] = 0;
    __syncthreads();

    // Part 1: Generate counts and masks
    unsigned int *sh_counts =& shmem[0];
    {
        Parameters reference, target;
        for ( unsigned int refIdx = threadIdx.x; refIdx < nSecs; refIdx += blockDim.x ) {
            reference.load(in_contrib + blockIdx.x * NPARAMS * nSecs + refIdx, nSecs);
            scalar norm = std::sqrt(reference.dotp(reference));
            unsigned int mask = compare_within_partition(reference, norm, dotp_threshold);
            unsigned int count = __popc(mask);
            out_masks[blockIdx.x * nPartitions * nSecs + (refIdx/32) * nSecs + refIdx] = mask;

            for ( int partitionOffset = 1; partitionOffset < nPartitions/2 + 1; partitionOffset++ ) {
                if ( (nPartitions&1) == 0 && partitionOffset == nPartitions/2 && (refIdx/32) >= nPartitions/2 ) {
                    // even # of data sets &&   it's the final iteration      && reference set is in the second half
                    // => This exact comparison has been done and recorded by the first half of reference sets on their final iteration.
                    break;
                }
                const int targetIdx = (refIdx + partitionOffset*32) % nSecs;
                target.load(in_contrib + blockIdx.x * NPARAMS * nSecs + targetIdx, nSecs);
                unsigned int target_mask;
                mask = compare_between_partitions(reference, target, norm, dotp_threshold, target_mask);
                count += __popc(mask);
                atomicAdd(&sh_counts[targetIdx], __popc(target_mask));
                out_masks[blockIdx.x * nPartitions * nSecs + (targetIdx/32) * nSecs + refIdx] = mask;
                out_masks[blockIdx.x * nPartitions * nSecs + (refIdx/32) * nSecs + targetIdx] = target_mask;
            }

            atomicAdd(&sh_counts[refIdx], count);
        }
        __syncthreads();
    }

    // Part 2: Find cluster head indices
    unsigned int static_headIdx;
    unsigned int nClusters;
    {
        unsigned int *sh_headIdx =& shmem[nSecs];
        for ( nClusters = 0; nClusters < MAXCLUSTERS; nClusters++ ) {
            // Block-stride reduction
            unsigned int headIdx = threadIdx.x;
            unsigned int headCount = sh_counts[headIdx];
            for ( unsigned int refIdx = threadIdx.x+blockDim.x; refIdx < nSecs; refIdx += blockDim.x ) {
                unsigned int count = sh_counts[refIdx];
                if ( count > headCount ) {
                    headCount = count;
                    headIdx = refIdx;
                }
            }

            // Warp reduction
            headIdx = warpReduceMaxIdx(headIdx, headCount);
            if ( laneid == 0 )
                sh_headIdx[warpid] = headIdx;
            __syncthreads();

            // Final reduction
            if ( warpid == 0 ) {
                headIdx = sh_headIdx[laneid];
                headCount = headIdx < nSecs ? sh_counts[headIdx] : 0;
                headIdx = warpReduceMaxIdx(headIdx, headCount);

                if ( laneid == 0 ) {
                    // Bail once cluster is too short
                    if ( sh_counts[headIdx] * secLen < minClusterLen )
                        headIdx = nSecs;
                    sh_headIdx[0] = headIdx;
                }
            }
            __syncthreads();

            // Read cluster head
            headIdx = sh_headIdx[0];
            if ( headIdx == nSecs ) // bail
                break;
            if ( threadIdx.x == nClusters )
                static_headIdx = headIdx;

            // Keep followers of head from being heads themselves
            for ( unsigned int secIdx = threadIdx.x; secIdx < nSecs; secIdx += blockDim.x ) {
                if ( out_masks[blockIdx.x * nPartitions * nSecs + (secIdx/32) * nSecs + headIdx] & (1 << (secIdx&31)) )
                    sh_counts[secIdx] = 0;
            }
            __syncthreads();
        }
    }

    // Part 3: Turn the head masks into timestamps
    unsigned int maxStops = shmem_size / nClusters;
    {
        if ( threadIdx.x < nClusters ) {
            unsigned int stopIdx = 1; // starts at 1 to allow space for the final stopIdx at 0
            for ( unsigned int partitionIdx = 0; partitionIdx < nPartitions && stopIdx < maxStops; partitionIdx++ ) {
                unsigned int mask = out_masks[blockIdx.x * nPartitions * nSecs + partitionIdx * nSecs + static_headIdx];
                for ( unsigned int i = 0; i < 32 && stopIdx < maxStops; i++ ) {
                    bool idle = (stopIdx&1); // No observation currently under way
                    bool mustsee = (mask & (1<<i)); // This bit should be included
                    if ( idle == mustsee ) {
                        shmem[threadIdx.x*maxStops + stopIdx] = 32*partitionIdx + i;
                        ++stopIdx;
                    }
                }
            }
            if ( !(stopIdx&1) && stopIdx < maxStops )
                shmem[threadIdx.x*maxStops + stopIdx++] = nPartitions*32-1;
            shmem[threadIdx.x*maxStops] = stopIdx-1;
        }
        __syncthreads();
    }

    // Part 4: Squeeze the timestamps into an iObservations, gather included current & deviations, and store the lot to output
    {
        unsigned int stopIdx;
        for ( unsigned int clusterIdx = warpid; clusterIdx < nClusters; clusterIdx += blockDim.x/32 ) {
            unsigned int nStops = shmem[clusterIdx*maxStops];

            // Shorten as necessary (note: each cluster dealt with in a single warp)
            while ( nStops > 2 * iObservations::maxObs ) {
                unsigned int shortestIdx = 0;
                unsigned int shortestStep = nSecs;
                unsigned int stepLen;

                // warp-stride reduce to find shortest step
                for ( unsigned int i = 0; i < (nStops+30)/31; i++ ) {
                    unsigned int tStop = 0;
                    stopIdx = 31*i + 1 + laneid;
                    if ( stopIdx < nStops )
                        tStop = shmem[clusterIdx*maxStops + stopIdx];
                    stepLen = __shfl_down_sync(0xffffffff, tStop, 1) - tStop;
                    if ( laneid < 31 && stopIdx < nStops && stepLen < shortestStep ) {
                        shortestStep = stepLen;
                        shortestIdx = stopIdx;
                    }
                }
                __syncwarp();

                // final reduce
                for ( unsigned int i = 1; i < warpSize; i *= 2 ) {
                    stepLen = __shfl_down_sync(0xffffffff, shortestStep, i);
                    stopIdx = __shfl_down_sync(0xffffffff, shortestIdx, i);
                    if ( stepLen < shortestStep ) {
                        shortestStep = stepLen;
                        shortestIdx = stopIdx;
                    }
                }
                shortestIdx = __shfl_sync(0xffffffff, shortestIdx, 0);

                // Shift all timestamps from shortestIdx+2 upwards down by two stops to eliminate the identified shorty
                nStops -= 2;
                for ( unsigned int i = shortestIdx/32; i <= nStops/32; i++ ) {
                    unsigned int tmp;
                    unsigned int idx = 32*i + laneid;
                    if ( idx >= shortestIdx && idx <= nStops )
                        tmp = shmem[clusterIdx*maxStops + idx + 2];
                    __syncwarp();
                    if ( idx >= shortestIdx && idx <= nStops )
                        shmem[clusterIdx*maxStops + idx] = tmp;
                }
            }

            // Gather current and deviation values across observed sections
            stopIdx = 0;
            scalar current = 0;
            int nAdditions = 0;
            Parameters contrib, tmp;
            contrib.zero();
            for ( unsigned int secIdx = laneid; secIdx < nSecs; secIdx += warpSize ) {
                while ( stopIdx < nStops && shmem[clusterIdx*maxStops + 1 + stopIdx] <= secIdx )
                    ++stopIdx;
                if ( stopIdx & 1 ) {
                    current += in_current[blockIdx.x * nSecs + secIdx];
                    tmp.load(in_contrib + blockIdx.x * NPARAMS * nSecs + secIdx, nSecs);
                    contrib += tmp;
                    ++nAdditions;
                }
            }
            __syncwarp();

            // Reduce into lane 0
            for ( unsigned int i = 1; i < warpSize; i *= 2 ) {
                current += __shfl_down_sync(0xffffffff, current, i);
                nAdditions += __shfl_down_sync(0xffffffff, nAdditions, i);
                tmp.shfl(contrib, laneid + i);
                contrib += tmp;
            }

            // Store output
            if ( laneid == 0 ) {
                iObservations obs = {{}, {}};
                for ( unsigned int i = 0; i < nStops/2; i++ ) {
                    obs.start[i] = shmem[clusterIdx*maxStops + 2*i + 1] * secLen;
                    obs.stop[i] = shmem[clusterIdx*maxStops + 2*i + 2] * secLen;
                }
                out_observations[blockIdx.x * MAXCLUSTERS + clusterIdx] = obs;

                contrib /= std::sqrt(contrib.dotp(contrib));
                contrib.store(out_clusters + blockIdx.x * MAXCLUSTERS * NPARAMS + clusterIdx * NPARAMS);

                current /= nAdditions;
                out_clusterCurrent[blockIdx.x * MAXCLUSTERS + clusterIdx] = current;
            }
        }

        // Backstop
        if ( nClusters < MAXCLUSTERS && threadIdx.x == 0 ) {
            out_observations[blockIdx.x * MAXCLUSTERS + nClusters] = iObservations {{},{}};
        }
    }
}

extern "C" void pullClusters(int nStims)
{
    CHECK_CUDA_ERRORS(hipMemcpy(clusters, d_clusters, nStims * MAXCLUSTERS * NPARAMS * sizeof(scalar), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERRORS(hipMemcpy(clusterCurrent, d_clusterCurrent, nStims * MAXCLUSTERS * sizeof(scalar), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERRORS(hipMemcpy(clusterObs, d_clusterObs, nStims * MAXCLUSTERS * sizeof(iObservations), hipMemcpyDeviceToHost));
}

extern "C" int pullPrimitives(int nStims, int duration, int secLen)
{
    int nSecs = (duration+secLen-1)/secLen;
    int nPartitions = (nSecs + 31)/32;
    CHECK_CUDA_ERRORS(hipMemcpy(sections, d_sections, nStims * nPartitions * NPARAMS * PARTITION_SIZE * sizeof(scalar), hipMemcpyDeviceToHost));
    return nPartitions * PARTITION_SIZE;
}

extern "C" int cluster(int trajLen, /* length of EE trajectory (power of 2, <=32) */
                       int nTraj, /* Number of EE trajectories */
                       int duration,
                       int secLen,
                       scalar dotp_threshold,
                       int minClusterLen,
                       std::vector<double> deltabar_arg,
                       bool pull_results)
{
    unsigned int nStims = NMODELS / (trajLen*nTraj);
    unsigned int nClusters = nStims * MAXCLUSTERS;
    int nSecs = (duration+secLen-1)/secLen;
    int nPartitions = (nSecs + 31)/32;

    resizeArrayPair(sections, d_sections, sections_size, nStims * nPartitions * NPARAMS * PARTITION_SIZE);
    resizeArray(d_currents, currents_size, nStims * nPartitions * PARTITION_SIZE);
    resizeArrayPair(clusters, d_clusters, clusters_size, nClusters * NPARAMS);
    resizeArray(d_clusterMasks, clusterMasks_size, nStims * nPartitions * 32*nPartitions);
    resizeArrayPair(clusterCurrent, d_clusterCurrent, clusterCurrent_size, nClusters);
    resizeArrayPair(clusterObs, d_clusterObs, clusterObs_size, nClusters);

    scalar deltabar_array[NPARAMS];
    for ( int i = 0; i < NPARAMS; i++ )
        deltabar_array[i] = deltabar_arg[i];
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(deltabar), deltabar_array, NPARAMS*sizeof(scalar)));

    dim3 block(STIMS_PER_CLUSTER_BLOCK * 32);
    dim3 grid(((nStims+STIMS_PER_CLUSTER_BLOCK-1)/STIMS_PER_CLUSTER_BLOCK));
    build_section_primitives<<<grid, block>>>(trajLen, nTraj, nStims, duration, secLen, nPartitions, d_sections, d_currents);

    size_t shmem_size = std::max(32*nPartitions, 8192);
    size_t nWarps = 16;
    exactClustering<<<nStims, 32*nWarps, shmem_size*sizeof(int)>>>(nPartitions, dotp_threshold, secLen, minClusterLen,
                                                                   d_sections, d_currents,
                                                                   d_clusters, d_clusterCurrent, d_clusterObs,
                                                                   d_clusterMasks, shmem_size);


    if ( pull_results )
        pullClusters(nStims);

    return nPartitions;
}





/// ******************************************************************************************************************************
///  >============================     Elementary Effects WG: Deltabar    ======================================================<
/// ******************************************************************************************************************************

static constexpr int STIMS_PER_DELTABAR_WARP = NPARAMS>16 ? 1 : NPARAMS>8 ? 2 : NPARAMS>4 ? 4 : NPARAMS>2 ? 8 : NPARAMS>1 ? 16 : 32;
static constexpr int STIMS_PER_DELTABAR_BLOCK = 8 * STIMS_PER_DELTABAR_WARP;

__global__ void find_deltabar_kernel(int trajLen, int nTraj, int nStims, int duration, scalar *global_clusters, int *global_clusterLen)
{
    static constexpr int stimWidth = 32/STIMS_PER_DELTABAR_WARP;
    const int nUsefulTraces = (trajLen-1)*nTraj;
    int paramIdx, stimIdx;
    if ( STIMS_PER_DELTABAR_WARP == 1 ) {
        // One row, one stim
        paramIdx = threadIdx.x;
        stimIdx = threadIdx.y;
    } else {
        // Each row is one warp with multiple stims
        paramIdx = threadIdx.x % stimWidth;
        stimIdx = (threadIdx.y * STIMS_PER_DELTABAR_WARP) + (threadIdx.x/stimWidth);
    }
    const int global_stimIdx = STIMS_PER_DELTABAR_BLOCK*blockIdx.x + stimIdx;
    const int id0 = global_stimIdx * trajLen * nTraj;
    const int nContrib = (nUsefulTraces/NPARAMS) + (paramIdx < (nUsefulTraces%NPARAMS));
    const iObservations obs = dd_obsUNI[id0];
    int nextObs = 0;
    int nSamples = 0;

    __shared__ scalar sh_clusters[STIMS_PER_DELTABAR_BLOCK][NPARAMS];
    __shared__ scalar sh_nSamples[STIMS_PER_DELTABAR_BLOCK];
    const int tid = threadIdx.y*blockDim.x + threadIdx.x;

    // Accumulate each stim's square deviations
    scalar sumSquares = 0;
    if ( paramIdx < NPARAMS ) {
        if ( global_stimIdx < nStims ) {
            for ( int t = 0; t < duration; t++ ) {
                if ( nextObs < iObservations::maxObs && t >= obs.start[nextObs] ) {
                    if ( t < obs.stop[nextObs] ) {
                        scalar contrib = 0;
                        for ( int i = paramIdx; i < nUsefulTraces; i += NPARAMS ) {
                            contrib += scalarfabs(dd_timeseries[t*NMODELS + id0 + i + i/(trajLen-1) + 1]);
                        }
                        contrib /= nContrib;
                        sumSquares += contrib*contrib;
                        ++nSamples;
                    } else {
                        ++nextObs;
                    }
                }
            }
        }
        sh_clusters[stimIdx][paramIdx] = sumSquares;
        if ( paramIdx == 0 )
            sh_nSamples[stimIdx] = nSamples;
    }

    // Reduce to a single 'cluster' in block
    for ( int width = STIMS_PER_DELTABAR_BLOCK; width > 1; width /= 32 ) {
        paramIdx = tid / width;
        stimIdx = tid % width;
        sumSquares = 0;
        nSamples = 0;
        __syncthreads();
        if ( paramIdx < NPARAMS ) {
            sumSquares = sh_clusters[stimIdx][paramIdx];
            if ( paramIdx == 0 )
                nSamples = sh_nSamples[stimIdx];
        }

        if ( width > 32 ) {
            sumSquares = warpReduceSum(sumSquares);
            if ( paramIdx == 0 )
                nSamples = warpReduceSum(nSamples);
            if ( stimIdx % 32 == 0 ) {
                sh_clusters[stimIdx/32][paramIdx] = sumSquares;
                if ( paramIdx == 0 )
                    sh_nSamples[stimIdx/32] = nSamples;
            }
        } else {
            sumSquares = warpReduceSum(sumSquares, width);
            if ( paramIdx == 0 )
                nSamples = warpReduceSum(nSamples, width);
        }
    }
    if ( stimIdx == 0 && paramIdx < NPARAMS ) {
        global_clusters[blockIdx.x*NPARAMS + paramIdx] = sumSquares;
        if ( paramIdx == 0 )
            global_clusterLen[blockIdx.x] = nSamples;
    }
}

extern "C" std::vector<double> find_deltabar(int trajLen, int nTraj, int duration)
{
    unsigned int nStims = NMODELS / (trajLen*nTraj);
    dim3 block(((NPARAMS+31)/32)*32, STIMS_PER_DELTABAR_BLOCK/STIMS_PER_DELTABAR_WARP);
    dim3 grid(((nStims+STIMS_PER_DELTABAR_BLOCK-1)/STIMS_PER_DELTABAR_BLOCK));

    resizeArrayPair(clusters, d_clusters, clusters_size, grid.x * NPARAMS);
    resizeArrayPair(clusterLen, d_clusterLen, clusterLen_size, grid.x);

    find_deltabar_kernel<<<grid, block>>>(trajLen, nTraj, nStims, duration, d_clusters, d_clusterLen);

    CHECK_CUDA_ERRORS(hipMemcpy(clusters, d_clusters, grid.x * NPARAMS * sizeof(scalar), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERRORS(hipMemcpy(clusterLen, d_clusterLen, grid.x * sizeof(int), hipMemcpyDeviceToHost));

    // Reduce across blocks on the CPU - this isn't performance-critical
    int n = 0;
    std::vector<double> ret(NPARAMS, 0);
    for ( int i = 0; i < grid.x; i++ ) {
        for ( int p = 0; p < NPARAMS; p++ )
            ret[p] += clusters[i*NPARAMS + p];
        n += clusterLen[i];
    }
    for ( int p = 0; p < NPARAMS; p++ )
        ret[p] = sqrt(ret[p] / n);
    return ret;
}





/// ******************************************************************************************************************************
///  >============================     Utility functions    ====================================================================<
/// ******************************************************************************************************************************

__global__ void observe_no_steps_kernel(int blankCycles)
{
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
    if ( id >= NMODELS )
        return;
    iStimulation stim = dd_stimUNI[id];
    iObservations obs = {};
    int tStart = 0;
    int nextObs = 0;
    for ( const auto step : stim ) {
        if ( step.t > stim.duration )
            break;
        if ( !step.ramp ) {
            if ( tStart < step.t ) {
                obs.start[nextObs] = tStart;
                obs.stop[nextObs] = step.t;
                if ( ++nextObs == iObservations::maxObs )
                    break;
            }
            tStart = step.t + blankCycles;
        }
    }
    if ( nextObs < iObservations::maxObs ) {
        if ( tStart < stim.duration ) {
            obs.start[nextObs] = tStart;
            obs.stop[nextObs] = stim.duration;
        }
    }
    dd_obsUNI[id] = obs;
}

extern "C" void observe_no_steps(int blankCycles)
{
    dim3 block(256);
    observe_no_steps_kernel<<<((NMODELS+block.x-1)/block.x)*block.x, block.x>>>(blankCycles);
}

#endif
