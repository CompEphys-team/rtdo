#include "hip/hip_runtime.h"
#ifndef UNIVERSAL_CU
#define UNIVERSAL_CU

#include "universallibrary.h"
#include "cuda_helper.h"

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/count.h>

static scalar *target = nullptr, *d_target = nullptr;
static __constant__ scalar *dd_target = nullptr;
static unsigned int target_size = 0, latest_target_size = 0;

static scalar *timeseries = nullptr, *d_timeseries = nullptr;
static __constant__ scalar *dd_timeseries = nullptr;
static unsigned int timeseries_size = 0, latest_timeseries_size = 0;

static __constant__ iStimulation singular_stim;
static __constant__ iObservations singular_obs;

static __constant__ scalar singular_clampGain;
static __constant__ scalar singular_accessResistance;
static __constant__ int singular_iSettleDuration;
static __constant__ scalar singular_Imax;
static __constant__ scalar singular_dt;

static __constant__ size_t singular_targetOffset;

// profiler memory space
static constexpr unsigned int NPAIRS = NMODELS/2;
static scalar *d_gradient;
static constexpr unsigned int gradientSz = NPAIRS * (NPAIRS - 1); // No diagonal

// elementary effects wg / clustering memory space
static scalar *clusters = nullptr, *d_clusters = nullptr;
static unsigned int clusters_size = 0;

static int *clusterLen = nullptr, *d_clusterLen = nullptr;
static unsigned int clusterLen_size = 0;

void libInit(UniversalLibrary &lib, UniversalLibrary::Pointers &pointers)
{
    pointers.pushV = [](void *hostptr, void *devptr, size_t size){
        CHECK_CUDA_ERRORS(hipMemcpy(devptr, hostptr, size, hipMemcpyHostToDevice))
    };
    pointers.pullV = [](void *hostptr, void *devptr, size_t size){
        CHECK_CUDA_ERRORS(hipMemcpy(hostptr, devptr, size, hipMemcpyDeviceToHost));
    };

    pointers.target =& target;
    pointers.output =& timeseries;

    pointers.clusters =& clusters;
    pointers.clusterLen =& clusterLen;

    allocateMem();
    initialize();

    hipGetSymbolAddress((void **)&lib.stim.singular_v, singular_stim);
    hipGetSymbolAddress((void **)&lib.obs.singular_v, singular_obs);

    hipGetSymbolAddress((void **)&lib.clampGain.singular_v, singular_clampGain);
    hipGetSymbolAddress((void **)&lib.accessResistance.singular_v, singular_accessResistance);
    hipGetSymbolAddress((void **)&lib.iSettleDuration.singular_v, singular_iSettleDuration);
    hipGetSymbolAddress((void **)&lib.Imax.singular_v, singular_Imax);
    hipGetSymbolAddress((void **)&lib.dt.singular_v, singular_dt);

    hipGetSymbolAddress((void **)&lib.targetOffset.singular_v, singular_targetOffset);

    CHECK_CUDA_ERRORS(hipMalloc(&d_gradient, gradientSz * sizeof(scalar)));
}

extern "C" void libExit(UniversalLibrary::Pointers &pointers)
{
    freeMem();
    pointers.pushV = pointers.pullV = nullptr;
}

extern "C" void resetDevice()
{
    hipDeviceReset();
}

extern "C" void resizeTarget(size_t newSize)
{
    resizeArrayPair(target, d_target, target_size, newSize);
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dd_target), &d_target, sizeof(scalar*)));
    latest_target_size = newSize;
}

extern "C" void pushTarget()
{
    CHECK_CUDA_ERRORS(hipMemcpy(d_target, target, latest_target_size * sizeof(scalar), hipMemcpyHostToDevice))
}

extern "C" void resizeOutput(size_t newSize)
{
    resizeArrayPair(timeseries, d_timeseries, timeseries_size, newSize);
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dd_timeseries), &d_timeseries, sizeof(scalar*)));
    latest_timeseries_size = newSize;
}

extern "C" void pullOutput()
{
    CHECK_CUDA_ERRORS(hipMemcpy(timeseries, d_timeseries, latest_timeseries_size * sizeof(scalar), hipMemcpyDeviceToHost))
}



/// ******************************************************************************************************************************
///  >============================     Profiler kernel & host function      ====================================================<
/// ******************************************************************************************************************************

// Compute the current deviation of both tuned and untuned models against each tuned model
// Models are interleaved (even id = tuned, odd id = detuned) in SamplingProfiler
__global__ void compute_gradient(int nSamples, int stride, scalar *targetParam, scalar *gradient)
{
    unsigned int xThread = blockIdx.x * blockDim.x + threadIdx.x; // probe
    unsigned int yThread = blockIdx.y * blockDim.y + threadIdx.y; // reference
    unsigned int x,y;
    if ( xThread < yThread ) { // transpose subdiagonal half of the top-left quadrant to run on the supradiagonal half of bottom-right quadrant
        // the coordinate transformation is equivalent to squashing the bottom-right supradiagonal triangle to the left border,
        // then flipping it up across the midline.
        x = xThread + NPAIRS - yThread; // xnew = x + n-y
        y = NPAIRS - yThread - 1;       // ynew = n-y - 1
    } else {
        x = xThread;
        y = yThread;
    }

    scalar err_tx_ty = 0., err_tx_dy = 0., err_dx_ty = 0., err;
    int i = 0;
    for ( ; i < nSamples; i += stride ) {
        scalar xval = dd_timeseries[2*x + NMODELS*i];
        scalar yval = dd_timeseries[2*y + NMODELS*i];

        err = xval - yval;
        err_tx_ty += err*err;

        err = xval - dd_timeseries[2*y+1 + NMODELS*i];
        err_tx_dy += err*err;

        err = yval - dd_timeseries[2*x+1 + NMODELS*i];
        err_dx_ty += err*err;
    }

    i = nSamples/stride; // Using i as nSamplesUsed
    err_tx_ty = std::sqrt(err_tx_ty / i);
    err_tx_dy = std::sqrt(err_tx_dy / i);
    err_dx_ty = std::sqrt(err_dx_ty / i);

    if ( x != y ) { // Ignore diagonal (don't probe against self)
        // invert sign as appropriate, such that detuning in the direction of the reference is reported as positive
        i = (1 - 2 * (targetParam[2*x] < targetParam[2*y])); // using i as sign

        // fractional change in error ( (d_err-t_err)/t_err) "how much does the error improve by detuning, relative to total error?")
        err = ((err_dx_ty / err_tx_ty) - 1) * i;

        // Put invalid values to the end of the scale, positive or negative; heuristically balance both sides
        if ( ::isnan(err) )
            err = i * SCALAR_MAX;

        // Addressing: Squish the diagonal out to prevent extra zeroes
        gradient[xThread + NPAIRS*yThread - yThread - (xThread>yThread)] = err;

        err = (1 - (err_tx_dy / err_tx_ty)) * i; // = ((err_tx_dy / err_tx_ty) - 1) * -i
        if ( ::isnan(err) )
            err = -i * SCALAR_MAX;
        gradient[xThread + NPAIRS*yThread - yThread - (xThread>yThread) + (NPAIRS-1)*(NPAIRS/2)] = err;
    }
}

struct is_positive : public thrust::unary_function<scalar, bool>
{
    __host__ __device__ bool operator()(scalar x){
        return x > 0;
    }
};

extern "C" void profile(int nSamples, int stride, scalar *d_targetParam, double &accuracy, double &median_norm_gradient)
{
    dim3 block(32, 16);
    dim3 grid(NPAIRS/32, NPAIRS/32);
    compute_gradient<<<grid, block>>>(nSamples, stride, d_targetParam, d_gradient);

    thrust::device_ptr<scalar> gradient = thrust::device_pointer_cast(d_gradient);
    thrust::sort(gradient, gradient + gradientSz);

    double nPositive = thrust::count_if(gradient, gradient + gradientSz, is_positive());
    accuracy = nPositive / gradientSz;

    scalar median_g[2];
    CHECK_CUDA_ERRORS(hipMemcpy(median_g, d_gradient + gradientSz/2, 2*sizeof(scalar), hipMemcpyDeviceToHost));
    median_norm_gradient = (median_g[0] + median_g[1]) / 2;
}





/// ******************************************************************************************************************************
///  >============================     Elementary Effects WG    ================================================================<
/// ******************************************************************************************************************************

static constexpr int STIMS_PER_CLUSTER_WARP = NPARAMS>16 ? 1 : NPARAMS>8 ? 2 : NPARAMS>4 ? 4 : NPARAMS>2 ? 8 : NPARAMS>1 ? 16 : 32;
static constexpr int STIMS_PER_CLUSTER_BLOCK = 8 * STIMS_PER_CLUSTER_WARP;

// Code adapated from Justin Luitjens, <https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/>
// Note, shuffle is only supported on compute capability 3.x and higher
__device__ inline scalar warpReduceSum(scalar val, int cutoff = warpSize)
{
    for ( int offset = 1; offset < cutoff; offset *= 2 )
        val += __shfl_down_sync(val, offset, 0xffffffff);
    return val;
}

__device__ inline scalar sumOverStim(scalar val, int stimWidth, int stimIdx)
{
    static __shared__ scalar tmp[STIMS_PER_CLUSTER_BLOCK];
    val = warpReduceSum(val, stimWidth);
    if ( NPARAMS > warpSize ) {
        if ( threadIdx.x & (stimWidth-1) == 0 ) // Lane 0
            atomicAdd(&tmp[stimIdx], val);
        __syncthreads();
        val = tmp[stimIdx];
    } else {
        val = __shfl_sync(val, 0, 0xffffffff);
    }
    return val;
}

__global__ void clusterKernel(int nTraces, /* total number of ee steps, a multiple of 31 */
                              int duration,
                              int secLen,
                              scalar dotp_threshold,
                              scalar *global_clusters, int *global_clusterLen)
{
    static constexpr int stimWidth = 32/STIMS_PER_CLUSTER_WARP;
    int paramIdx, stimIdx;
    if ( STIMS_PER_CLUSTER_WARP == 1 ) {
        // One row, one stim
        paramIdx = threadIdx.x;
        stimIdx = threadIdx.y;
    } else {
        // Each row is one warp with multiple stims
        paramIdx = threadIdx.x % stimWidth;
        stimIdx = (threadIdx.y * STIMS_PER_CLUSTER_WARP) + (threadIdx.x/stimWidth);
    }
    int global_stimIdx = STIMS_PER_CLUSTER_BLOCK*blockIdx.x + stimIdx;
    int timeseries_offset = global_stimIdx * (nTraces/31)*32;

    __shared__ scalar sh_clusters[STIMS_PER_CLUSTER_BLOCK][MAXCLUSTERS][NPARAMS];
    __shared__ scalar sh_cluster_square_norm[STIMS_PER_CLUSTER_BLOCK][MAXCLUSTERS];
    __shared__ int sh_clusterLen[STIMS_PER_CLUSTER_BLOCK][MAXCLUSTERS];
    { // zero-initialise
        if ( paramIdx < NPARAMS )
            for ( int i = 0; i < MAXCLUSTERS; i++ )
                sh_clusters[stimIdx][i][paramIdx] = 0;

        int laneid = threadIdx.x + blockDim.x*threadIdx.y;
        if ( laneid < STIMS_PER_CLUSTER_BLOCK ) {
            for ( int i = 0; i < MAXCLUSTERS; i++ ) {
                sh_cluster_square_norm[laneid][i] = 0;
                sh_clusterLen[laneid][i] = 0;
            }
        }
    }

    // Construct clusters in shared memory
    scalar contrib, square;
    for ( int t = 0; t < duration; t++ ) {

        // Load ee contribution from this parameter
        // Note, timeseries[0,32,64,...] are unused (COMPARE_PREVTHREAD), hence "+ i/31 + 1" indexing
        contrib = 0;
        if ( paramIdx < NPARAMS )
            for ( int tEnd = (t + secLen > duration) ? duration : (t + secLen); t < tEnd; t++ )
                for ( int i = paramIdx; i < nTraces; i += NPARAMS )
                    contrib += dd_timeseries[t*NMODELS + timeseries_offset + i + i/31 + 1];

        // Compute the square norm (sum of squared contributions) for scalar product normalisation
        square = contrib * contrib;
        square = sumOverStim(square, stimWidth, stimIdx);

        // Compute the scalar product with each existing cluster to find the closest match
        scalar max_dotp = 0;
        int closest_cluster = 0;
        int nClusters = 0;
        for ( int i = 0; i < MAXCLUSTERS; ++i ) {
            scalar dotp = 0;

            // Ignore empty sections (all zeroes is (a) useless and (b) likely in an unobserved region)
            if ( square > 0 ) {
                if ( sh_clusterLen[stimIdx][i] > 0 ) {
                    ++nClusters;
                    if ( paramIdx < NPARAMS )
                        dotp = contrib * sh_clusters[stimIdx][i][paramIdx];
                }
                dotp = sumOverStim(dotp, stimWidth, stimIdx);
                dotp /= std::sqrt(square * sh_cluster_square_norm[stimIdx][i]); // normalised
                if ( (max_dotp >= 0 && dotp > max_dotp) || (max_dotp < 0 && dotp < max_dotp) ) {
                    max_dotp = dotp;
                    closest_cluster = i;
                }
            } else if ( NPARAMS > warpSize ) {
                // sumOverStim has a __syncthreads call, idle over it to prevent stalling
                dotp = sumOverStim(dotp, stimWidth, stimIdx);
            }
        }

        // No adequate cluster: start a new one
        if ( scalarfabs(max_dotp) < dotp_threshold ) {
            closest_cluster = nClusters;
        }

        // Add present contribution to the nearest cluster, ignoring empty sections and cluster overflow
        if ( square > 0 && closest_cluster < MAXCLUSTERS ) {
            if ( paramIdx < NPARAMS ) {
                contrib += sh_clusters[stimIdx][closest_cluster][paramIdx] * (max_dotp < 0 ? -1 : 1);
                sh_clusters[stimIdx][closest_cluster][paramIdx] = contrib;
            }
            square = contrib * contrib;
            square = sumOverStim(square, stimWidth, stimIdx);
            if ( paramIdx == 0 ) {
                sh_cluster_square_norm[stimIdx][closest_cluster] = square;
                ++sh_clusterLen[stimIdx][closest_cluster];
            }
        }
        __syncthreads();
    }

    // Push normalised completed clusters to global memory
    if ( paramIdx < NPARAMS ) {
        for ( int i = 0; i < MAXCLUSTERS; i++ ) {
            contrib = sh_clusters[stimIdx][i][paramIdx];
            square = contrib*contrib;
            global_clusters[(((global_stimIdx * MAXCLUSTERS) + i) * NPARAMS) + paramIdx] =
                    square / sh_cluster_square_norm[stimIdx][i];
            if ( paramIdx == 0 ) {
                global_clusterLen[(global_stimIdx * MAXCLUSTERS) + i] = sh_clusterLen[stimIdx][i];
            }
        }
    }
}

extern "C" void cluster(int nTraces, /* total number of ee steps, a multiple of 31 */
             int duration,
             int secLen,
             scalar dotp_threshold)
{
    unsigned int nStims = NMODELS / ((nTraces/31)*32);
    unsigned int nClusters = nStims * MAXCLUSTERS;
    resizeArrayPair(clusters, d_clusters, clusters_size, nClusters * NPARAMS);
    resizeArrayPair(clusterLen, d_clusterLen, clusterLen_size, nClusters);

    dim3 block((NPARAMS+31)/32, STIMS_PER_CLUSTER_BLOCK/STIMS_PER_CLUSTER_WARP);
    dim3 grid(nStims);
    clusterKernel<<<grid, block>>>(nTraces, duration, secLen, dotp_threshold, d_clusters, d_clusterLen);

    CHECK_CUDA_ERRORS(hipMemcpy(clusters, d_clusters, nClusters * NPARAMS * sizeof(scalar), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERRORS(hipMemcpy(clusterLen, d_clusterLen, nClusters * sizeof(int), hipMemcpyDeviceToHost));
}

#endif
