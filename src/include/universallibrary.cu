#include "hip/hip_runtime.h"
#ifndef UNIVERSAL_CU
#define UNIVERSAL_CU

#include "universallibrary.h"
#include "cuda_helper.h"

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/count.h>

static scalar *target = nullptr, *d_target = nullptr;
static __constant__ scalar *dd_target = nullptr;
static unsigned int target_size = 0, latest_target_size = 0;

static scalar *timeseries = nullptr, *d_timeseries = nullptr;
static __constant__ scalar *dd_timeseries = nullptr;
static unsigned int timeseries_size = 0, latest_timeseries_size = 0;

static __constant__ iStimulation singular_stim;
static __constant__ iObservations singular_obs;

static __constant__ scalar singular_clampGain;
static __constant__ scalar singular_accessResistance;
static __constant__ int singular_iSettleDuration;
static __constant__ scalar singular_Imax;
static __constant__ scalar singular_dt;

static __constant__ size_t singular_targetOffset;

// profiler memory space
static constexpr unsigned int NPAIRS = NMODELS/2;
static scalar *d_gradient;
static constexpr unsigned int gradientSz = NPAIRS * (NPAIRS - 1); // No diagonal

// elementary effects wg / clustering memory space
static scalar *clusters = nullptr, *d_clusters = nullptr;
static unsigned int clusters_size = 0;

static int *clusterLen = nullptr, *d_clusterLen = nullptr;
static unsigned int clusterLen_size = 0;

static unsigned int *d_clusterMasks = nullptr;
static unsigned int clusterMasks_size = 0;

static scalar *clusterCurrent = nullptr, *d_clusterCurrent = nullptr;
static unsigned int clusterCurrent_size = 0;

static scalar *sections = nullptr, *d_sections = nullptr;
static unsigned int sections_size = 0;

static scalar *d_currents= nullptr;
static unsigned int currents_size = 0;

static iObservations *clusterObs = nullptr, *d_clusterObs = nullptr;
static unsigned int clusterObs_size = 0;

static Bubble *bubbles = nullptr, *d_bubbles = nullptr;
static unsigned int bubbles_size = 0;

static __constant__ scalar deltabar[NPARAMS];

void libInit(UniversalLibrary &lib, UniversalLibrary::Pointers &pointers)
{
    pointers.pushV = [](void *hostptr, void *devptr, size_t size){
        CHECK_CUDA_ERRORS(hipMemcpy(devptr, hostptr, size, hipMemcpyHostToDevice))
    };
    pointers.pullV = [](void *hostptr, void *devptr, size_t size){
        CHECK_CUDA_ERRORS(hipMemcpy(hostptr, devptr, size, hipMemcpyDeviceToHost));
    };

    pointers.target =& target;
    pointers.output =& timeseries;

    pointers.clusters =& clusters;
    pointers.clusterCurrent =& clusterCurrent;
    pointers.clusterPrimitives =& sections;
    pointers.clusterObs =& clusterObs;

    pointers.bubbles =& bubbles;

    allocateMem();
    initialize();

    hipGetSymbolAddress((void **)&lib.stim.singular_v, singular_stim);
    hipGetSymbolAddress((void **)&lib.obs.singular_v, singular_obs);

    hipGetSymbolAddress((void **)&lib.clampGain.singular_v, singular_clampGain);
    hipGetSymbolAddress((void **)&lib.accessResistance.singular_v, singular_accessResistance);
    hipGetSymbolAddress((void **)&lib.iSettleDuration.singular_v, singular_iSettleDuration);
    hipGetSymbolAddress((void **)&lib.Imax.singular_v, singular_Imax);
    hipGetSymbolAddress((void **)&lib.dt.singular_v, singular_dt);

    hipGetSymbolAddress((void **)&lib.targetOffset.singular_v, singular_targetOffset);

    CHECK_CUDA_ERRORS(hipMalloc(&d_gradient, gradientSz * sizeof(scalar)));
}

extern "C" void libExit(UniversalLibrary::Pointers &pointers)
{
    freeMem();
    pointers.pushV = pointers.pullV = nullptr;
}

extern "C" void resetDevice()
{
    hipDeviceReset();
}

extern "C" void resizeTarget(size_t newSize)
{
    resizeArrayPair(target, d_target, target_size, newSize);
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dd_target), &d_target, sizeof(scalar*)));
    latest_target_size = newSize;
}

extern "C" void pushTarget()
{
    CHECK_CUDA_ERRORS(hipMemcpy(d_target, target, latest_target_size * sizeof(scalar), hipMemcpyHostToDevice))
}

extern "C" void resizeOutput(size_t newSize)
{
    resizeArrayPair(timeseries, d_timeseries, timeseries_size, newSize);
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dd_timeseries), &d_timeseries, sizeof(scalar*)));
    latest_timeseries_size = newSize;
}

extern "C" void pullOutput()
{
    CHECK_CUDA_ERRORS(hipMemcpy(timeseries, d_timeseries, latest_timeseries_size * sizeof(scalar), hipMemcpyDeviceToHost))
}


template <typename T>
__device__ inline T warpReduceSum(T val, int cutoff = warpSize)
{
    for ( int offset = 1; offset < cutoff; offset *= 2 )
        val += __shfl_xor_sync(0xffffffff, val, offset);
    return val;
}

template <>
__device__ inline Parameters warpReduceSum<Parameters>(Parameters val, int cutoff)
{
    Parameters addend;
    for ( int offset = 1; offset < cutoff; offset *= 2 ) {
        addend.shfl(val, (threadIdx.x&31) ^ offset, cutoff);
        val += addend;
    }
    return val;
}



/// ******************************************************************************************************************************
///  >============================     Profiler kernel & host function      ====================================================<
/// ******************************************************************************************************************************

// Compute the current deviation of both tuned and untuned models against each tuned model
// Models are interleaved (even id = tuned, odd id = detuned) in SamplingProfiler
__global__ void compute_gradient(int nSamples, int stride, scalar *targetParam, scalar *gradient)
{
    unsigned int xThread = blockIdx.x * blockDim.x + threadIdx.x; // probe
    unsigned int yThread = blockIdx.y * blockDim.y + threadIdx.y; // reference
    unsigned int x,y;
    if ( xThread < yThread ) { // transpose subdiagonal half of the top-left quadrant to run on the supradiagonal half of bottom-right quadrant
        // the coordinate transformation is equivalent to squashing the bottom-right supradiagonal triangle to the left border,
        // then flipping it up across the midline.
        x = xThread + NPAIRS - yThread; // xnew = x + n-y
        y = NPAIRS - yThread - 1;       // ynew = n-y - 1
    } else {
        x = xThread;
        y = yThread;
    }

    scalar err_tx_ty = 0., err_tx_dy = 0., err_dx_ty = 0., err;
    int i = 0;
    for ( ; i < nSamples; i += stride ) {
        scalar xval = dd_timeseries[2*x + NMODELS*i];
        scalar yval = dd_timeseries[2*y + NMODELS*i];

        err = xval - yval;
        err_tx_ty += err*err;

        err = xval - dd_timeseries[2*y+1 + NMODELS*i];
        err_tx_dy += err*err;

        err = yval - dd_timeseries[2*x+1 + NMODELS*i];
        err_dx_ty += err*err;
    }

    i = nSamples/stride; // Using i as nSamplesUsed
    err_tx_ty = std::sqrt(err_tx_ty / i);
    err_tx_dy = std::sqrt(err_tx_dy / i);
    err_dx_ty = std::sqrt(err_dx_ty / i);

    if ( x != y ) { // Ignore diagonal (don't probe against self)
        // invert sign as appropriate, such that detuning in the direction of the reference is reported as positive
        i = (1 - 2 * (targetParam[2*x] < targetParam[2*y])); // using i as sign

        // fractional change in error ( (d_err-t_err)/t_err) "how much does the error improve by detuning, relative to total error?")
        err = ((err_dx_ty / err_tx_ty) - 1) * i;

        // Put invalid values to the end of the scale, positive or negative; heuristically balance both sides
        if ( ::isnan(err) )
            err = i * SCALAR_MAX;

        // Addressing: Squish the diagonal out to prevent extra zeroes
        gradient[xThread + NPAIRS*yThread - yThread - (xThread>yThread)] = err;

        err = (1 - (err_tx_dy / err_tx_ty)) * i; // = ((err_tx_dy / err_tx_ty) - 1) * -i
        if ( ::isnan(err) )
            err = -i * SCALAR_MAX;
        gradient[xThread + NPAIRS*yThread - yThread - (xThread>yThread) + (NPAIRS-1)*(NPAIRS/2)] = err;
    }
}

struct is_positive : public thrust::unary_function<scalar, bool>
{
    __host__ __device__ bool operator()(scalar x){
        return x > 0;
    }
};

extern "C" void profile(int nSamples, int stride, scalar *d_targetParam, double &accuracy, double &median_norm_gradient)
{
    dim3 block(32, 16);
    dim3 grid(NPAIRS/32, NPAIRS/32);
    compute_gradient<<<grid, block>>>(nSamples, stride, d_targetParam, d_gradient);

    thrust::device_ptr<scalar> gradient = thrust::device_pointer_cast(d_gradient);
    thrust::sort(gradient, gradient + gradientSz);

    double nPositive = thrust::count_if(gradient, gradient + gradientSz, is_positive());
    accuracy = nPositive / gradientSz;

    scalar median_g[2];
    CHECK_CUDA_ERRORS(hipMemcpy(median_g, d_gradient + gradientSz/2, 2*sizeof(scalar), hipMemcpyDeviceToHost));
    median_norm_gradient = (median_g[0] + median_g[1]) / 2;
}





/// ******************************************************************************************************************************
///  >============================     Elementary Effects WG: Clustering    ====================================================<
/// ******************************************************************************************************************************
static constexpr int STIMS_PER_CLUSTER_BLOCK = 16;
static constexpr int PARTITION_SIZE = 32;

/**
 * @brief build_section_primitives chops the EE traces in d_timeseries into deltabar-normalised deviation vectors ("sections")
 *          representing up to secLen ticks. Sections are chunked into partitions of PARTITION_SIZE=32 sections each.
 *          Deviation vectors represent the mean deviation per tick, normalised to deltabar, caused by a single detuning.
 *          Note, this kernel expects the EE traces to be generated using TIMESERIES_COMPARE_NONE
 * @param out_sections is the output, laid out as [stimIdx][paramIdx][partitionIdx][secIdx (local to partition)].
 * @param out_current is the mean current within each section, laid out as [stimIdx][partitionIdx][secIdx].
 */
__global__ void build_section_primitives(const int trajLen,
                                         const int nTraj,
                                         const int nStims,
                                         const int duration,
                                         const int secLen,
                                         const int nPartitions,
                                         scalar *out_sections,
                                         scalar *out_current)
{
    const int warpid = threadIdx.x / 32; // acts as block-local stim idx
    const int laneid = threadIdx.x & 31;
    const int stimIdx = (blockIdx.x * blockDim.x + threadIdx.x) / 32; // global stim idx; one stim per warp
    const int nTraces = trajLen*nTraj; // Total number of traces per stim, including starting point models
    const int nUsefulTraces = (trajLen-1)*nTraj; // Number of mid-trajectory models per stim
    const int paramIdx_after_end_of_final_traj = nUsefulTraces % NPARAMS; // First param idx with one contrib fewer than the preceding ones
    const int lane0_offset = stimIdx * nTraces;
    const int nLoads = (nTraces + 31) & 0xffffffe0;
    const iObservations obs = dd_obsUNI[stimIdx*nTraces];
    int nextObs = 0;

    volatile __shared__ scalar sh_contrib[STIMS_PER_CLUSTER_BLOCK][NPARAMS][PARTITION_SIZE + 1];
    volatile __shared__ scalar sh_current[STIMS_PER_CLUSTER_BLOCK][PARTITION_SIZE + 1];

    for ( int i = threadIdx.x; i < STIMS_PER_CLUSTER_BLOCK*NPARAMS*(PARTITION_SIZE+1); i += blockDim.x )
        *((scalar*)sh_contrib + i) = 0;
    for ( int i = threadIdx.x; i < STIMS_PER_CLUSTER_BLOCK*(PARTITION_SIZE+1); i += blockDim.x )
        *((scalar*)sh_current + i) = 0;
    __syncthreads();

    if ( stimIdx >= nStims )
        return;

    unsigned int secIdx = 0;
    int trueSecLen_static;
    int t = 0;
    while ( t < duration ) {
        int trueSecLen = 0;
        __syncwarp();
        for ( int tEnd = t + secLen; t < tEnd; t++ ) { // Note, t<duration guaranteed by obs.stop
            if ( nextObs < iObservations::maxObs && t >= obs.start[nextObs] ) {
                if ( t < obs.stop[nextObs] ) {
                    for ( int i = laneid; i < nLoads; i += warpSize ) {
                        const int paramIdx = (i - 1 - (i/trajLen)) % NPARAMS;
                        scalar current_mylane = dd_timeseries[t*NMODELS + lane0_offset + i];
                        scalar current_prevlane = __shfl_up_sync(0xffffffff, current_mylane, 1);
                        scalar diff = scalarfabs(current_prevlane - current_mylane);
                        if ( i < nTraces ) {
                            if ( i % trajLen != 0 )
                                atomicAdd((scalar*)&sh_contrib[warpid][paramIdx][secIdx&31], diff);
                            current_mylane = scalarfabs(current_mylane);
                        } else {
                            current_mylane = 0;
                        }
                        current_mylane = warpReduceSum(current_mylane);
                        if ( laneid == 0 )
                            sh_current[warpid][secIdx&31] += current_mylane;
                    }
                    ++trueSecLen;
                } else {
                    ++nextObs;
                }
            }
        }
        if ( laneid == (secIdx&31) )
            trueSecLen_static = trueSecLen;

        if ( ((++secIdx) & 31) == 0 || t >= duration ) {
            __syncwarp();
            const int partitionIdx = (secIdx-1) >> 5;
            int nContrib = nUsefulTraces/NPARAMS + 1;
            if ( t < duration || laneid <= (secIdx&31) ) {
                for ( int paramIdx = 0; paramIdx < NPARAMS; paramIdx++ ) {
                    if ( paramIdx == paramIdx_after_end_of_final_traj )
                        --nContrib;
                    out_sections[stimIdx * NPARAMS * nPartitions * PARTITION_SIZE
                            + paramIdx * nPartitions * PARTITION_SIZE
                            + partitionIdx * PARTITION_SIZE
                            + laneid]
                            = trueSecLen_static
                              ? sh_contrib[warpid][paramIdx][laneid] / (trueSecLen_static * deltabar[paramIdx] * nContrib)
                              : 0;
                    sh_contrib[warpid][paramIdx][laneid] = 0;
                }
            }

            out_current[stimIdx * nPartitions * PARTITION_SIZE
                    + partitionIdx * PARTITION_SIZE
                    + laneid]
                    = sh_current[warpid][laneid] / (trueSecLen_static * nTraces);
            sh_current[warpid][laneid] = 0;
        }
    }
}

/**
 * @brief compare_within_partition compares all sections in a partition to each other, recording a similarity for each
 * @param myContrib is a section's deviation vector
 * @param dotp_threshold
 * @return a bitmask flagging each above-threshold similar section
 */
__device__ unsigned int compare_within_partition(const Parameters myContrib,
                                                 const scalar norm,
                                                 const scalar dotp_threshold)
{
    const unsigned laneid = threadIdx.x & 31;
    unsigned int mask = 1<<laneid;
    Parameters target_contrib;

    for ( int offset = 1; offset < 17; offset++ ) {
        int target = (laneid + offset)&31;

        // Compare against target
        target_contrib.shfl(myContrib, target);
        scalar target_norm = __shfl_sync(0xffffffff, norm, target);
        scalar dotp = myContrib.dotp(target_contrib);
        if ( dotp > 0 )
            dotp /= (norm * target_norm);

        // Process my own work
        if ( dotp > dotp_threshold ) {
            mask |= 1 << target;
        }

        // Retrieve the work of the thread that targeted me, and process that, too
        target = (laneid + 32 - offset)&31;
        dotp = __shfl_sync(0xffffffff, dotp, target);
        if ( offset < 16 && dotp > dotp_threshold ) {
            mask |= 1 << target;
        }
    }
    return mask;
}

/**
 * @brief compare_within_partition compares all sections in a partition to each other, recording a similarity for each
 * @param myContrib is a section's deviation vector
 * @param dotp_threshold
 * @return a bitmask flagging each above-threshold similar section
 */
__device__ unsigned int compare_between_partitions(const Parameters reference,
                                                   Parameters target,
                                                   const scalar ref_norm,
                                                   const scalar dotp_threshold,
                                                   unsigned int &target_mask)
{
    const unsigned laneid = threadIdx.x & 31;
    unsigned int ref_mask = 0;
    target_mask = 0;
    scalar target_norm = std::sqrt(target.dotp(target));
    const int srcLane = (laneid+1) & 31;
    for ( int i = 0; i < warpSize; i++ ) {
        // Compare against target
        scalar dotp = reference.dotp(target);
        if ( dotp > 0 )
            dotp /= (ref_norm * target_norm);
        if ( dotp > dotp_threshold ) {
            // Update reference
            ref_mask |= 1 << ((laneid+i)&31);

            // Update target
            target_mask |= 1 << laneid;
        }

        // Shuffle targets down (except after the final comparison)
        if ( i < 31 ) {
            target.shfl(target, srcLane);
            target_norm = __shfl_sync(0xffffffff, target_norm, srcLane);
        }
        // shuffle target mask down 32 times to return it to its original lane
        target_mask = __shfl_sync(0xffffffff, target_mask, srcLane);
    }
    return ref_mask;
}

template <typename T>
__device__ unsigned int warpReduceMaxIdx(unsigned int idx, T value)
{
    for ( unsigned int i = 1; i < warpSize; i *= 2 ) {
        T cmp_value = __shfl_down_sync(0xffffffff, value, i);
        unsigned int cmp_idx = __shfl_down_sync(0xffffffff, idx, i);
        if ( cmp_value > value ) {
            value = cmp_value;
            idx = cmp_idx;
        }
    }
    return __shfl_sync(0xffffffff, idx, 0);
}

/**
 * @brief exactClustering is a non-heuristic clustering implementation. It takes the outputs from build_section_primitives, extracting
 * for each stim a set of clusters with associated iObservations, Euclidean normal deviation vector, and mean current.
 */
__global__ void exactClustering(const int nPartitions,
                                const scalar dotp_threshold,
                                const int secLen,
                                const int minClusterLen,
                                scalar *in_contrib, /* [stimIdx][paramIdx][secIdx] */
                                scalar *in_current, /* [stimIdx][secIdx] */
                                scalar *out_clusters, /* [stimIdx][clusterIdx][paramIdx] */
                                scalar *out_clusterCurrent, /* [stimIdx][clusterIdx] */
                                iObservations *out_observations, /* [stimIdx][clusterIdx] */
                                unsigned int *out_masks, /* [stimIdx][partitionIdx][secIdx], intermediate only */
                                const unsigned int shmem_size /* in uints. Minimum nSecs+32, preferably much more for obs timestamp leeway */
                                )
{
    const unsigned laneid = threadIdx.x & 31;
    const unsigned warpid = threadIdx.x >> 5;
    const unsigned int nSecs = 32 * nPartitions;

    extern __shared__ unsigned int shmem[];
    for ( int i = threadIdx.x; i < shmem_size; i += blockDim.x )
        shmem[i] = 0;
    __syncthreads();

    // Part 1: Generate counts and masks
    unsigned int *sh_counts =& shmem[0];
    {
        Parameters reference, target;
        for ( unsigned int refIdx = threadIdx.x; refIdx < nSecs; refIdx += blockDim.x ) {
            reference.load(in_contrib + blockIdx.x * NPARAMS * nSecs + refIdx, nSecs);
            scalar norm = std::sqrt(reference.dotp(reference));
            unsigned int mask = compare_within_partition(reference, norm, dotp_threshold);
            unsigned int count = __popc(mask);
            out_masks[blockIdx.x * nPartitions * nSecs + (refIdx/32) * nSecs + refIdx] = mask;

            for ( int partitionOffset = 1; partitionOffset < nPartitions/2 + 1; partitionOffset++ ) {
                if ( (nPartitions&1) == 0 && partitionOffset == nPartitions/2 && (refIdx/32) >= nPartitions/2 ) {
                    // even # of data sets &&   it's the final iteration      && reference set is in the second half
                    // => This exact comparison has been done and recorded by the first half of reference sets on their final iteration.
                    break;
                }
                const int targetIdx = (refIdx + partitionOffset*32) % nSecs;
                target.load(in_contrib + blockIdx.x * NPARAMS * nSecs + targetIdx, nSecs);
                unsigned int target_mask;
                mask = compare_between_partitions(reference, target, norm, dotp_threshold, target_mask);
                count += __popc(mask);
                atomicAdd(&sh_counts[targetIdx], __popc(target_mask));
                out_masks[blockIdx.x * nPartitions * nSecs + (targetIdx/32) * nSecs + refIdx] = mask;
                out_masks[blockIdx.x * nPartitions * nSecs + (refIdx/32) * nSecs + targetIdx] = target_mask;
            }

            atomicAdd(&sh_counts[refIdx], count);
        }
        __syncthreads();
    }

    // Part 2: Find cluster head indices
    unsigned int static_headIdx;
    unsigned int nClusters;
    {
        unsigned int *sh_headIdx =& shmem[nSecs];
        for ( nClusters = 0; nClusters < MAXCLUSTERS; nClusters++ ) {
            // Block-stride reduction
            unsigned int headIdx = threadIdx.x;
            unsigned int headCount = sh_counts[headIdx];
            for ( unsigned int refIdx = threadIdx.x+blockDim.x; refIdx < nSecs; refIdx += blockDim.x ) {
                unsigned int count = sh_counts[refIdx];
                if ( count > headCount ) {
                    headCount = count;
                    headIdx = refIdx;
                }
            }

            // Warp reduction
            headIdx = warpReduceMaxIdx(headIdx, headCount);
            if ( laneid == 0 )
                sh_headIdx[warpid] = headIdx;
            __syncthreads();

            // Final reduction
            if ( warpid == 0 ) {
                headIdx = sh_headIdx[laneid];
                headCount = headIdx < nSecs ? sh_counts[headIdx] : 0;
                headIdx = warpReduceMaxIdx(headIdx, headCount);

                if ( laneid == 0 ) {
                    // Bail once cluster is too short
                    if ( sh_counts[headIdx] * secLen < minClusterLen )
                        headIdx = nSecs;
                    sh_headIdx[0] = headIdx;
                }
            }
            __syncthreads();

            // Read cluster head
            headIdx = sh_headIdx[0];
            if ( headIdx == nSecs ) // bail
                break;
            if ( threadIdx.x == nClusters )
                static_headIdx = headIdx;

            // Keep followers of head from being heads themselves
            for ( unsigned int secIdx = threadIdx.x; secIdx < nSecs; secIdx += blockDim.x ) {
                if ( out_masks[blockIdx.x * nPartitions * nSecs + (secIdx/32) * nSecs + headIdx] & (1 << (secIdx&31)) )
                    sh_counts[secIdx] = 0;
            }
            __syncthreads();
        }
    }

    // Part 3: Turn the head masks into timestamps
    unsigned int maxStops = shmem_size / nClusters;
    {
        if ( threadIdx.x < nClusters ) {
            unsigned int stopIdx = 1; // starts at 1 to allow space for the final stopIdx at 0
            for ( unsigned int partitionIdx = 0; partitionIdx < nPartitions && stopIdx < maxStops; partitionIdx++ ) {
                unsigned int mask = out_masks[blockIdx.x * nPartitions * nSecs + partitionIdx * nSecs + static_headIdx];
                for ( unsigned int i = 0; i < 32 && stopIdx < maxStops; i++ ) {
                    bool idle = (stopIdx&1); // No observation currently under way
                    bool mustsee = (mask & (1<<i)); // This bit should be included
                    if ( idle == mustsee ) {
                        shmem[threadIdx.x*maxStops + stopIdx] = 32*partitionIdx + i;
                        ++stopIdx;
                    }
                }
            }
            if ( !(stopIdx&1) && stopIdx < maxStops )
                shmem[threadIdx.x*maxStops + stopIdx++] = nPartitions*32-1;
            shmem[threadIdx.x*maxStops] = stopIdx-1;
        }
        __syncthreads();
    }

    // Part 4: Squeeze the timestamps into an iObservations, gather included current & deviations, and store the lot to output
    {
        unsigned int stopIdx;
        for ( unsigned int clusterIdx = warpid; clusterIdx < nClusters; clusterIdx += blockDim.x/32 ) {
            unsigned int nStops = shmem[clusterIdx*maxStops];

            // Shorten as necessary (note: each cluster dealt with in a single warp)
            while ( nStops > 2 * iObservations::maxObs ) {
                unsigned int shortestIdx = 0;
                unsigned int shortestStep = nSecs;
                unsigned int stepLen;

                // warp-stride reduce to find shortest step
                for ( unsigned int i = 0; i < (nStops+30)/31; i++ ) {
                    unsigned int tStop = 0;
                    stopIdx = 31*i + 1 + laneid;
                    if ( stopIdx < nStops )
                        tStop = shmem[clusterIdx*maxStops + stopIdx];
                    stepLen = __shfl_down_sync(0xffffffff, tStop, 1) - tStop;
                    if ( laneid < 31 && stopIdx < nStops && stepLen < shortestStep ) {
                        shortestStep = stepLen;
                        shortestIdx = stopIdx;
                    }
                }
                __syncwarp();

                // final reduce
                for ( unsigned int i = 1; i < warpSize; i *= 2 ) {
                    stepLen = __shfl_down_sync(0xffffffff, shortestStep, i);
                    stopIdx = __shfl_down_sync(0xffffffff, shortestIdx, i);
                    if ( stepLen < shortestStep ) {
                        shortestStep = stepLen;
                        shortestIdx = stopIdx;
                    }
                }
                shortestIdx = __shfl_sync(0xffffffff, shortestIdx, 0);

                // Shift all timestamps from shortestIdx+2 upwards down by two stops to eliminate the identified shorty
                nStops -= 2;
                for ( unsigned int i = shortestIdx/32; i <= nStops/32; i++ ) {
                    unsigned int tmp;
                    unsigned int idx = 32*i + laneid;
                    if ( idx >= shortestIdx && idx <= nStops )
                        tmp = shmem[clusterIdx*maxStops + idx + 2];
                    __syncwarp();
                    if ( idx >= shortestIdx && idx <= nStops )
                        shmem[clusterIdx*maxStops + idx] = tmp;
                }
            }

            // Gather current and deviation values across observed sections
            stopIdx = 0;
            scalar current = 0;
            int nAdditions = 0;
            Parameters contrib, tmp;
            contrib.zero();
            for ( unsigned int secIdx = laneid; secIdx < nSecs; secIdx += warpSize ) {
                while ( stopIdx < nStops && shmem[clusterIdx*maxStops + 1 + stopIdx] <= secIdx )
                    ++stopIdx;
                if ( stopIdx & 1 ) {
                    current += in_current[blockIdx.x * nSecs + secIdx];
                    tmp.load(in_contrib + blockIdx.x * NPARAMS * nSecs + secIdx, nSecs);
                    contrib += tmp;
                    ++nAdditions;
                }
            }
            __syncwarp();

            // Reduce into lane 0
            for ( unsigned int i = 1; i < warpSize; i *= 2 ) {
                current += __shfl_down_sync(0xffffffff, current, i);
                nAdditions += __shfl_down_sync(0xffffffff, nAdditions, i);
                tmp.shfl(contrib, laneid + i);
                contrib += tmp;
            }

            // Store output
            if ( laneid == 0 ) {
                iObservations obs = {{}, {}};
                for ( unsigned int i = 0; i < nStops/2; i++ ) {
                    obs.start[i] = shmem[clusterIdx*maxStops + 2*i + 1] * secLen;
                    obs.stop[i] = shmem[clusterIdx*maxStops + 2*i + 2] * secLen;
                }
                out_observations[blockIdx.x * MAXCLUSTERS + clusterIdx] = obs;

                contrib /= std::sqrt(contrib.dotp(contrib));
                contrib.store(out_clusters + blockIdx.x * MAXCLUSTERS * NPARAMS + clusterIdx * NPARAMS);

                current /= nAdditions;
                out_clusterCurrent[blockIdx.x * MAXCLUSTERS + clusterIdx] = current;
            }
        }

        // Backstop
        if ( nClusters < MAXCLUSTERS && threadIdx.x == 0 ) {
            out_observations[blockIdx.x * MAXCLUSTERS + nClusters] = iObservations {{},{}};
        }
    }
}

extern "C" void pullClusters(int nStims)
{
    CHECK_CUDA_ERRORS(hipMemcpy(clusters, d_clusters, nStims * MAXCLUSTERS * NPARAMS * sizeof(scalar), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERRORS(hipMemcpy(clusterCurrent, d_clusterCurrent, nStims * MAXCLUSTERS * sizeof(scalar), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERRORS(hipMemcpy(clusterObs, d_clusterObs, nStims * MAXCLUSTERS * sizeof(iObservations), hipMemcpyDeviceToHost));
}

extern "C" int pullPrimitives(int nStims, int duration, int secLen)
{
    int nSecs = (duration+secLen-1)/secLen;
    int nPartitions = (nSecs + 31)/32;
    CHECK_CUDA_ERRORS(hipMemcpy(sections, d_sections, nStims * nPartitions * NPARAMS * PARTITION_SIZE * sizeof(scalar), hipMemcpyDeviceToHost));
    return nPartitions * PARTITION_SIZE;
}

extern "C" void cluster(int trajLen, /* length of EE trajectory (power of 2, <=32) */
                       int nTraj, /* Number of EE trajectories */
                       int duration,
                       int secLen,
                       scalar dotp_threshold,
                       int minClusterLen,
                       std::vector<double> deltabar_arg,
                       bool pull_results)
{
    unsigned int nStims = NMODELS / (trajLen*nTraj);
    unsigned int nClusters = nStims * MAXCLUSTERS;
    int nSecs = (duration+secLen-1)/secLen;
    int nPartitions = (nSecs + 31)/32;

    resizeArrayPair(sections, d_sections, sections_size, nStims * nPartitions * NPARAMS * PARTITION_SIZE);
    resizeArray(d_currents, currents_size, nStims * nPartitions * PARTITION_SIZE);
    resizeArrayPair(clusters, d_clusters, clusters_size, nClusters * NPARAMS);
    resizeArray(d_clusterMasks, clusterMasks_size, nStims * nPartitions * 32*nPartitions);
    resizeArrayPair(clusterCurrent, d_clusterCurrent, clusterCurrent_size, nClusters);
    resizeArrayPair(clusterObs, d_clusterObs, clusterObs_size, nClusters);

    scalar deltabar_array[NPARAMS];
    for ( int i = 0; i < NPARAMS; i++ )
        deltabar_array[i] = deltabar_arg[i];
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(deltabar), deltabar_array, NPARAMS*sizeof(scalar)));

    dim3 block(STIMS_PER_CLUSTER_BLOCK * 32);
    dim3 grid(((nStims+STIMS_PER_CLUSTER_BLOCK-1)/STIMS_PER_CLUSTER_BLOCK));
    build_section_primitives<<<grid, block>>>(trajLen, nTraj, nStims, duration, secLen, nPartitions, d_sections, d_currents);

    size_t shmem_size = std::max(32*nPartitions, 8192);
    size_t nWarps = 16;
    exactClustering<<<nStims, 32*nWarps, shmem_size*sizeof(int)>>>(nPartitions, dotp_threshold, secLen, minClusterLen,
                                                                   d_sections, d_currents,
                                                                   d_clusters, d_clusterCurrent, d_clusterObs,
                                                                   d_clusterMasks, shmem_size);


    if ( pull_results )
        pullClusters(nStims);
}






/// ******************************************************************************************************************************
///  >============================     Elementary Effects WG: Bubbles    =======================================================<
/// ******************************************************************************************************************************

__device__ inline Bubble bubble_shfl(Bubble b, int srcLane)
{
    return Bubble {
        __shfl_sync(0xffffffff, b.startCycle, srcLane),
        __shfl_sync(0xffffffff, b.cycles, srcLane),
        __shfl_sync(0xffffffff, b.value, srcLane)
    };
}

/**
 * @brief warpMergeBubbles merges bubble triplets across a warp, storing in lane 0's bubbles the fully merged triplet
 * @param cyclesPerTriplet Number of cycles (sections?) that each triplet summarises
 * @param cutoff Number of threads participating in the merge (power of 2, or pass zero for other threads' triplets. All threads must call this function in sync.)
 */
__device__ void warpMergeBubbles(Bubble &start, Bubble &mid, Bubble &end, int cyclesPerTriplet, int cutoff)
{
    Bubble nextStart, nextMid, nextEnd;
    const unsigned int laneid = threadIdx.x&31;
    for ( int offset = 1; offset < cutoff; offset *= 2 ) {
        // Shuffle down
        nextStart = bubble_shfl(start, laneid+offset);
        nextMid = bubble_shfl(mid, laneid+offset);
        nextEnd = bubble_shfl(end, laneid+offset);

        // Merge across the seam
        if ( nextStart.cycles ) {
            if ( start.cycles == cyclesPerTriplet*offset ) { // full-length merge
                // Merge nextStart into start; invalidate nextStart
                start.cycles += nextStart.cycles;
                start.value += nextStart.value;
                nextStart.cycles = 0;
            } else if ( end.cycles ) { // end merge
                if ( nextStart.cycles == cyclesPerTriplet*offset ) { // nextStart is full-length
                    // Merge end and nextStart, save to new end, invalidate end and nextStart
                    nextEnd = {end.startCycle,
                               end.cycles+nextStart.cycles,
                               end.value+nextStart.value};
                    end.cycles = 0;
                } else {
                    // Merge nextStart into end; invalidate nextStart
                    end.cycles += nextStart.cycles;
                    end.value += nextStart.value;
                }
                nextStart.cycles = 0;
            } else if ( nextStart.cycles == cyclesPerTriplet*offset ) { // No merge, nextStart is new end
                nextEnd = nextStart;
                nextStart.cycles = 0;
            }
        }

        // Replace mid with the largest of (mid, end, nextStart, nextMid)
        scalar midFitness = mid.cycles ? mid.value/mid.cycles : 0;
        if ( end.cycles && end.value/end.cycles > midFitness ) {
            midFitness = end.value/end.cycles;
            mid = end;
        }
        if ( nextStart.cycles && nextStart.value/nextStart.cycles > midFitness ) {
            midFitness = nextStart.value/nextStart.cycles;
            mid = nextStart;
        }
        if ( nextMid.cycles && nextMid.value/nextMid.cycles > midFitness )
            mid = nextMid;

        // Replace end with nextEnd
        end = nextEnd;
    }
}

__global__ void buildBubbles(const int nPartitions,
                             const int secLen,
                             scalar *in_contrib, /* [stimIdx][paramIdx][secIdx] */
                             scalar *in_current, /* [stimIdx][secIdx] */
                             scalar *out_deviations, /* [stimIdx][targetParamIdx][paramIdx] */
                             scalar *out_bubbleCurrents, /* [stimIdx][targetParamIdx] */
                             Bubble *out_bubbles /* [stimIdx][targetParamIdx] */
                             )
{
    const unsigned int nSecs = 32*nPartitions;
    const unsigned int warpid = threadIdx.x/32;
    const unsigned int laneid = threadIdx.x&31;
    const unsigned int nWarps = blockDim.x/32;
    const unsigned int stimIdx = blockIdx.x;
    const unsigned int targetParamIdx = blockIdx.y;

    Bubble start, mid, end;
    Parameters dev;
    extern __shared__ unsigned int shmem[];
    Bubble *sh_start = (Bubble*) &shmem[0];
    Bubble *sh_mid =& sh_start[nPartitions];
    Bubble *sh_end =& sh_mid[nPartitions];

    // Generate bubble triplets over partitions
    for ( unsigned int partitionIdx = warpid; partitionIdx < nPartitions; partitionIdx += nWarps ) {
        const unsigned int secIdx = partitionIdx*32 + laneid;
        // Generate initial bubble triplet over section
        dev.load(in_contrib + stimIdx * NPARAMS * nSecs + secIdx, nSecs);
        start.value = dev[targetParamIdx] / dev.mean();
        start.startCycle = secIdx;
        start.cycles = (start.value > 1);
        mid = {0,0,0};
        end = {0,0,0};

        // Merge to lane 0
        warpMergeBubbles(start, mid, end, 1, warpSize);

        if ( laneid == 0 ) {
            sh_start[partitionIdx] = start;
            sh_mid[partitionIdx] = mid;
            sh_end[partitionIdx] = end;
        }
    }

    __syncthreads();

    // Reduce across partitions in warp 0
    if ( warpid == 0 ) {
        Bubble prevStart, prevMid, prevEnd;
        for ( unsigned int metaPartitionIdx = 0; metaPartitionIdx < (nPartitions+31)/32; metaPartitionIdx++ ) {
            const unsigned int partitionIdx = metaPartitionIdx*32 + laneid;
            // Load partition triplet
            if ( partitionIdx < nPartitions ) {
                start = sh_start[partitionIdx];
                mid = sh_mid[partitionIdx];
                end = sh_end[partitionIdx];
            } else {
                start = mid = end = {0,0,0};
            }

            // Merge and broadcast
            warpMergeBubbles(start, mid, end, warpSize, warpSize);
            start = bubble_shfl(start, 0);
            mid = bubble_shfl(mid, 0);
            end = bubble_shfl(end, 0);

            // Merge the previous MetaPartition's triplet in lane 0 with the current one in lane 1
            if ( metaPartitionIdx > 0 ) {
                if ( laneid == 0 ) {
                    start = prevStart;
                    mid = prevMid;
                    end = prevEnd;
                }
                warpMergeBubbles(start, mid, end, 1024, 2);
            }
            if ( laneid == 0 ) {
                prevStart = start;
                prevMid = mid;
                prevEnd = end;
            }
        }

        // Pick the winner, store to shmem and to output
        if ( laneid == 0 ) {
            // Assume mid to be the fittest (likeliest branch)
            scalar fitness = mid.cycles ? mid.value/mid.cycles : 0;
            if ( start.cycles && start.value/start.cycles > fitness ) {
                fitness = start.value/start.cycles;
                mid = start;
            }
            if ( end.cycles && end.value/end.cycles > fitness ) {
                fitness = end.value/end.cycles;
                mid = end;
            }
            mid.value = fitness;
            sh_start[0] = mid;

            // Adjust timings from secs to ticks
            mid.cycles *= secLen;
            mid.startCycle *= secLen;
            out_bubbles[stimIdx * NPARAMS + targetParamIdx] = mid;
        }
    }

    __syncthreads();
    start = sh_start[0];
    __syncthreads();

    // Discard use of shmem as Bubble[], use it for current/deviation reduction instead
    // Note the size requirement for 32 scalars and 32 Parameters.
    scalar *sh_current = (scalar*)&shmem[0];
    Parameters *sh_deviation = (Parameters*)&sh_current[32];
    if ( laneid == 0 ) {
        sh_current[warpid] = 0;
        sh_deviation[warpid].zero();
    }

    // Gather bubble deviation and current
    for ( unsigned int secIdx = start.startCycle, lastSec = start.startCycle + start.cycles; secIdx < (lastSec+31)&0xffffffe0; secIdx += blockDim.x ) {
        scalar current;
        if ( secIdx < lastSec ) {
            dev.load(in_contrib + stimIdx * NPARAMS * nSecs + secIdx, nSecs);
            current = in_current[stimIdx * nSecs + secIdx];
        } else {
            dev.zero();
            current = 0;
        }

        current = warpReduceSum(current);
        dev = warpReduceSum(dev);

        if ( laneid == 0 ) {
            sh_current[warpid] += current;
            sh_deviation[warpid] += dev;
        }
    }

    __syncthreads();

    if ( warpid == 0 ) {
        scalar current = warpReduceSum(sh_current[laneid]);
        dev = warpReduceSum(sh_deviation[laneid]);
        if ( laneid == 0 ) {
            out_bubbleCurrents[stimIdx * NPARAMS + targetParamIdx] = current;
            dev /= std::sqrt(dev.dotp(dev));
            dev.store(out_deviations + stimIdx * NPARAMS * NPARAMS + targetParamIdx * NPARAMS);
        }
    }
}

extern "C" void pullBubbles(int nStims)
{
    CHECK_CUDA_ERRORS(hipMemcpy(bubbles, d_bubbles, nStims * NPARAMS * sizeof(Bubble), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERRORS(hipMemcpy(clusterCurrent, d_clusterCurrent, nStims * NPARAMS * sizeof(scalar), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERRORS(hipMemcpy(clusters, d_clusters, nStims * NPARAMS * NPARAMS * sizeof(scalar), hipMemcpyDeviceToHost));
}

extern "C" void bubble(int trajLen, /* length of EE trajectory (power of 2, <=32) */
                       int nTraj, /* Number of EE trajectories */
                       int duration,
                       int secLen,
                       std::vector<double> deltabar_arg,
                       bool pull_results)
{
    unsigned int nStims = NMODELS / (trajLen*nTraj);
    int nSecs = (duration+secLen-1)/secLen;
    int nPartitions = (nSecs + 31)/32;

    resizeArrayPair(sections, d_sections, sections_size, nStims * nPartitions * NPARAMS * PARTITION_SIZE);
    resizeArray(d_currents, currents_size, nStims * nPartitions * PARTITION_SIZE);
    resizeArrayPair(clusters, d_clusters, clusters_size, nStims * NPARAMS * NPARAMS);
    resizeArrayPair(clusterCurrent, d_clusterCurrent, clusterCurrent_size, nStims * NPARAMS);
    resizeArrayPair(bubbles, d_bubbles, bubbles_size, nStims * NPARAMS);

    scalar deltabar_array[NPARAMS];
    for ( int i = 0; i < NPARAMS; i++ )
        deltabar_array[i] = deltabar_arg[i];
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(deltabar), deltabar_array, NPARAMS*sizeof(scalar)));

    dim3 block(STIMS_PER_CLUSTER_BLOCK * 32);
    dim3 grid(((nStims+STIMS_PER_CLUSTER_BLOCK-1)/STIMS_PER_CLUSTER_BLOCK));
    build_section_primitives<<<grid, block>>>(trajLen, nTraj, nStims, duration, secLen, nPartitions, d_sections, d_currents);

    size_t nWarps = 16;
    size_t shmem_for_bubbles = 3 * nPartitions * sizeof(Bubble);
    size_t shmem_for_stats = 32 * sizeof(scalar) + 32 * sizeof(Parameters);
    size_t shmem_size = std::max(shmem_for_bubbles, shmem_for_stats);
    block = dim3(nWarps * 32);
    grid = dim3(nStims, NPARAMS);
    buildBubbles<<<grid, block, shmem_size>>>(nPartitions, secLen, d_sections, d_currents, d_clusters, d_clusterCurrent, d_bubbles);

    if ( pull_results )
        pullBubbles(nStims);
}





/// ******************************************************************************************************************************
///  >============================     Elementary Effects WG: Deltabar    ======================================================<
/// ******************************************************************************************************************************

static constexpr int STIMS_PER_DELTABAR_WARP = NPARAMS>16 ? 1 : NPARAMS>8 ? 2 : NPARAMS>4 ? 4 : NPARAMS>2 ? 8 : NPARAMS>1 ? 16 : 32;
static constexpr int STIMS_PER_DELTABAR_BLOCK = 8 * STIMS_PER_DELTABAR_WARP;

__global__ void find_deltabar_kernel(int trajLen, int nTraj, int nStims, int duration, scalar *global_clusters, int *global_clusterLen)
{
    static constexpr int stimWidth = 32/STIMS_PER_DELTABAR_WARP;
    const int nUsefulTraces = (trajLen-1)*nTraj;
    int paramIdx, stimIdx;
    if ( STIMS_PER_DELTABAR_WARP == 1 ) {
        // One row, one stim
        paramIdx = threadIdx.x;
        stimIdx = threadIdx.y;
    } else {
        // Each row is one warp with multiple stims
        paramIdx = threadIdx.x % stimWidth;
        stimIdx = (threadIdx.y * STIMS_PER_DELTABAR_WARP) + (threadIdx.x/stimWidth);
    }
    const int global_stimIdx = STIMS_PER_DELTABAR_BLOCK*blockIdx.x + stimIdx;
    const int id0 = global_stimIdx * trajLen * nTraj;
    const int nContrib = (nUsefulTraces/NPARAMS) + (paramIdx < (nUsefulTraces%NPARAMS));
    const iObservations obs = dd_obsUNI[id0];
    int nextObs = 0;
    int nSamples = 0;

    __shared__ scalar sh_clusters[STIMS_PER_DELTABAR_BLOCK][NPARAMS];
    __shared__ scalar sh_nSamples[STIMS_PER_DELTABAR_BLOCK];
    const int tid = threadIdx.y*blockDim.x + threadIdx.x;

    // Accumulate each stim's square deviations
    scalar sumSquares = 0;
    if ( paramIdx < NPARAMS ) {
        if ( global_stimIdx < nStims ) {
            for ( int t = 0; t < duration; t++ ) {
                if ( nextObs < iObservations::maxObs && t >= obs.start[nextObs] ) {
                    if ( t < obs.stop[nextObs] ) {
                        scalar contrib = 0;
                        for ( int i = paramIdx; i < nUsefulTraces; i += NPARAMS ) {
                            contrib += scalarfabs(dd_timeseries[t*NMODELS + id0 + i + i/(trajLen-1) + 1]);
                        }
                        contrib /= nContrib;
                        sumSquares += contrib*contrib;
                        ++nSamples;
                    } else {
                        ++nextObs;
                    }
                }
            }
        }
        sh_clusters[stimIdx][paramIdx] = sumSquares;
        if ( paramIdx == 0 )
            sh_nSamples[stimIdx] = nSamples;
    }

    // Reduce to a single 'cluster' in block
    for ( int width = STIMS_PER_DELTABAR_BLOCK; width > 1; width /= 32 ) {
        paramIdx = tid / width;
        stimIdx = tid % width;
        sumSquares = 0;
        nSamples = 0;
        __syncthreads();
        if ( paramIdx < NPARAMS ) {
            sumSquares = sh_clusters[stimIdx][paramIdx];
            if ( paramIdx == 0 )
                nSamples = sh_nSamples[stimIdx];
        }

        if ( width > 32 ) {
            sumSquares = warpReduceSum(sumSquares);
            if ( paramIdx == 0 )
                nSamples = warpReduceSum(nSamples);
            if ( stimIdx % 32 == 0 ) {
                sh_clusters[stimIdx/32][paramIdx] = sumSquares;
                if ( paramIdx == 0 )
                    sh_nSamples[stimIdx/32] = nSamples;
            }
        } else {
            sumSquares = warpReduceSum(sumSquares, width);
            if ( paramIdx == 0 )
                nSamples = warpReduceSum(nSamples, width);
        }
    }
    if ( stimIdx == 0 && paramIdx < NPARAMS ) {
        global_clusters[blockIdx.x*NPARAMS + paramIdx] = sumSquares;
        if ( paramIdx == 0 )
            global_clusterLen[blockIdx.x] = nSamples;
    }
}

extern "C" std::vector<double> find_deltabar(int trajLen, int nTraj, int duration)
{
    unsigned int nStims = NMODELS / (trajLen*nTraj);
    dim3 block(((NPARAMS+31)/32)*32, STIMS_PER_DELTABAR_BLOCK/STIMS_PER_DELTABAR_WARP);
    dim3 grid(((nStims+STIMS_PER_DELTABAR_BLOCK-1)/STIMS_PER_DELTABAR_BLOCK));

    resizeArrayPair(clusters, d_clusters, clusters_size, grid.x * NPARAMS);
    resizeArrayPair(clusterLen, d_clusterLen, clusterLen_size, grid.x);

    find_deltabar_kernel<<<grid, block>>>(trajLen, nTraj, nStims, duration, d_clusters, d_clusterLen);

    CHECK_CUDA_ERRORS(hipMemcpy(clusters, d_clusters, grid.x * NPARAMS * sizeof(scalar), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERRORS(hipMemcpy(clusterLen, d_clusterLen, grid.x * sizeof(int), hipMemcpyDeviceToHost));

    // Reduce across blocks on the CPU - this isn't performance-critical
    int n = 0;
    std::vector<double> ret(NPARAMS, 0);
    for ( int i = 0; i < grid.x; i++ ) {
        for ( int p = 0; p < NPARAMS; p++ )
            ret[p] += clusters[i*NPARAMS + p];
        n += clusterLen[i];
    }
    for ( int p = 0; p < NPARAMS; p++ )
        ret[p] = sqrt(ret[p] / n);
    return ret;
}





/// ******************************************************************************************************************************
///  >============================     Utility functions    ====================================================================<
/// ******************************************************************************************************************************

__global__ void observe_no_steps_kernel(int blankCycles)
{
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
    if ( id >= NMODELS )
        return;
    iStimulation stim = dd_stimUNI[id];
    iObservations obs = {};
    int tStart = 0;
    int nextObs = 0;
    for ( const auto step : stim ) {
        if ( step.t > stim.duration )
            break;
        if ( !step.ramp ) {
            if ( tStart < step.t ) {
                obs.start[nextObs] = tStart;
                obs.stop[nextObs] = step.t;
                if ( ++nextObs == iObservations::maxObs )
                    break;
            }
            tStart = step.t + blankCycles;
        }
    }
    if ( nextObs < iObservations::maxObs ) {
        if ( tStart < stim.duration ) {
            obs.start[nextObs] = tStart;
            obs.stop[nextObs] = stim.duration;
        }
    }
    dd_obsUNI[id] = obs;
}

extern "C" void observe_no_steps(int blankCycles)
{
    dim3 block(256);
    observe_no_steps_kernel<<<((NMODELS+block.x-1)/block.x)*block.x, block.x>>>(blankCycles);
}

#endif
