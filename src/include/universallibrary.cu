#ifndef EXPERIMENT_CU
#define EXPERIMENT_CU

#include "universallibrary.h"
#include "cuda_helper.h"

static scalar *target = nullptr, *d_target = nullptr;
static __device__ scalar *dd_target = nullptr;
static unsigned int target_size = 0, latest_target_size = 0;

static scalar *timeseries = nullptr, *d_timeseries = nullptr;
static __device__ scalar *dd_timeseries = nullptr;
static unsigned int timeseries_size = 0, latest_timeseries_size = 0;

void libInit(UniversalLibrary::Pointers &pointers, size_t numModels)
{
    pointers.pushV = [numModels](void *hostptr, void *devptr, size_t size){
        CHECK_CUDA_ERRORS(hipMemcpy(devptr, hostptr, numModels * size, hipMemcpyHostToDevice))
    };
    pointers.pullV = [numModels](void *hostptr, void *devptr, size_t size){
        CHECK_CUDA_ERRORS(hipMemcpy(hostptr, devptr, numModels * size, hipMemcpyDeviceToHost));
    };

    pointers.target =& target;
    pointers.output =& timeseries;

    allocateMem();
    initialize();
}

extern "C" void libExit(UniversalLibrary::Pointers &pointers)
{
    freeMem();
    pointers.pushV = pointers.pullV = nullptr;
}

extern "C" void resetDevice()
{
    hipDeviceReset();
}

extern "C" void resizeTarget(size_t newSize)
{
    unsigned int tmp = target_size;
    resizeHostArray(target, tmp, newSize);
    resizeArray(d_target, target_size, newSize);
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dd_target), &d_target, sizeof(scalar*)));
    latest_target_size = newSize;
}

extern "C" void pushTarget()
{
    CHECK_CUDA_ERRORS(hipMemcpy(d_target, target, latest_target_size * sizeof(scalar), hipMemcpyHostToDevice))
}

extern "C" void resizeOutput(size_t newSize)
{
    unsigned int tmp = target_size;
    resizeHostArray(timeseries, tmp, newSize);
    resizeArray(d_timeseries, timeseries_size, newSize);
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dd_timeseries), &d_timeseries, sizeof(scalar*)));
    latest_timeseries_size = newSize;
}

extern "C" void pullOutput()
{
    CHECK_CUDA_ERRORS(hipMemcpy(timeseries, d_timeseries, latest_timeseries_size * sizeof(scalar), hipMemcpyDeviceToHost))
}

#endif
