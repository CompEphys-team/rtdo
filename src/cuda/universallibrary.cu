#ifndef UNIVERSAL_CU
#define UNIVERSAL_CU

#include "lib_definitions.h"

#include "bubble.cu"
#include "cluster.cu"
#include "deltabar.cu"
#include "deviations.cu"
#include "profile.cu"
#include "util.cu"

static std::vector<hipStream_t> lib_streams(1, 0);
inline hipStream_t getLibStream(unsigned int streamId)
{
    unsigned int oldSz = lib_streams.size();
    if ( streamId >= oldSz ) {
        lib_streams.resize(streamId+1);
        for ( unsigned int i = oldSz; i < streamId+1; i++ )
            hipStreamCreate(&lib_streams[i]);
    }
    return lib_streams[streamId];
}

static std::vector<hipEvent_t> lib_events;
static unsigned int nextEvent = 0;
inline hipEvent_t getLibEvent(unsigned int eventHandle)
{
    unsigned int oldSz = lib_events.size();
    if ( eventHandle >= oldSz ) {
        lib_events.resize(eventHandle+1);
        for ( unsigned int i = oldSz; i < eventHandle+1; i++ )
            hipEventCreate(&lib_events[i]);
    }
    return lib_events[eventHandle];
}

void libInit(UniversalLibrary &lib, UniversalLibrary::Pointers &pointers)
{
    pointers.pushV = [](void *hostptr, void *devptr, size_t size, int streamId){
        if ( streamId < 0 )
            CHECK_CUDA_ERRORS(hipMemcpy(devptr, hostptr, size, hipMemcpyHostToDevice))
        else
            CHECK_CUDA_ERRORS(hipMemcpyAsync(devptr, hostptr, size, hipMemcpyHostToDevice, getLibStream(streamId)))
    };
    pointers.pullV = [](void *hostptr, void *devptr, size_t size, int streamId){
        if ( streamId < 0 )
            CHECK_CUDA_ERRORS(hipMemcpy(hostptr, devptr, size, hipMemcpyDeviceToHost))
        else
            CHECK_CUDA_ERRORS(hipMemcpyAsync(hostptr, devptr, size, hipMemcpyDeviceToHost, getLibStream(streamId)))
    };

    pointers.target =& target;
    pointers.output =& timeseries;
    pointers.summary =& summary;

    pointers.clusters =& clusters;
    pointers.clusterCurrent =& clusterCurrent;
    pointers.clusterPrimitives =& sections;
    pointers.clusterObs =& clusterObs;

    pointers.bubbles =& bubbles;

    allocateMem();
    initialize();

    hipGetSymbolAddress((void **)&lib.stim.singular_v, singular_stim);
    hipGetSymbolAddress((void **)&lib.obs.singular_v, singular_obs);

    hipGetSymbolAddress((void **)&lib.clampGain.singular_v, singular_clampGain);
    hipGetSymbolAddress((void **)&lib.accessResistance.singular_v, singular_accessResistance);
    hipGetSymbolAddress((void **)&lib.iSettleDuration.singular_v, singular_iSettleDuration);
    hipGetSymbolAddress((void **)&lib.Imax.singular_v, singular_Imax);
    hipGetSymbolAddress((void **)&lib.dt.singular_v, singular_dt);

    hipGetSymbolAddress((void **)&lib.targetOffset.singular_v, singular_targetOffset);

    CHECK_CUDA_ERRORS(hipMalloc(&d_prof_error, profSz * sizeof(scalar)));
    CHECK_CUDA_ERRORS(hipMalloc(&d_prof_dist_uw, profSz * sizeof(scalar)));
    CHECK_CUDA_ERRORS(hipMalloc(&d_prof_dist_to, profSz * sizeof(scalar)));
    CHECK_CUDA_ERRORS(hipMalloc(&d_prof_dist_w, profSz * sizeof(scalar)));

    // Philox is fastest for normal dist, if developer.nvidia.com/hiprand is to be believed
    CURAND_CALL(hiprandCreateGenerator(&cuRNG, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
}

extern "C" void libExit(UniversalLibrary::Pointers &pointers)
{
    for ( size_t i = 1; i < lib_streams.size(); i++ )
        CHECK_CUDA_ERRORS(hipStreamDestroy(lib_streams[i]));

    freeMem();
    pointers.pushV = pointers.pullV = nullptr;
    CURAND_CALL(hiprandDestroyGenerator(cuRNG));

    CHECK_CUDA_ERRORS(hipFree(d_target));
    CHECK_CUDA_ERRORS(hipFree(d_timeseries));
    CHECK_CUDA_ERRORS(hipFree(d_summary));
    CHECK_CUDA_ERRORS(hipFree(d_prof_error));
    CHECK_CUDA_ERRORS(hipFree(d_prof_dist_uw));
    CHECK_CUDA_ERRORS(hipFree(d_prof_dist_to));
    CHECK_CUDA_ERRORS(hipFree(d_prof_dist_w));
    CHECK_CUDA_ERRORS(hipFree(d_random));
    CHECK_CUDA_ERRORS(hipFree(d_clusters));
    CHECK_CUDA_ERRORS(hipFree(d_clusterLen));
    CHECK_CUDA_ERRORS(hipFree(d_clusterMasks));
    CHECK_CUDA_ERRORS(hipFree(d_clusterCurrent));
    CHECK_CUDA_ERRORS(hipFree(d_sections));
    CHECK_CUDA_ERRORS(hipFree(d_currents));
    CHECK_CUDA_ERRORS(hipFree(d_clusterObs));
    CHECK_CUDA_ERRORS(hipFree(d_bubbles));

    CHECK_CUDA_ERRORS(hipHostFree(target));
    CHECK_CUDA_ERRORS(hipHostFree(timeseries));
    CHECK_CUDA_ERRORS(hipHostFree(summary));
    CHECK_CUDA_ERRORS(hipHostFree(clusters));
    CHECK_CUDA_ERRORS(hipHostFree(clusterLen));
    CHECK_CUDA_ERRORS(hipHostFree(clusterCurrent));
    CHECK_CUDA_ERRORS(hipHostFree(sections));
    CHECK_CUDA_ERRORS(hipHostFree(clusterObs));
    CHECK_CUDA_ERRORS(hipHostFree(bubbles));
}

extern "C" void resetDevice()
{
    hipDeviceReset();
}

extern "C" void resizeTarget(size_t newSize)
{
    resizeArrayPair(target, d_target, target_size, newSize);
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dd_target), &d_target, sizeof(scalar*)));
    latest_target_size = newSize;
}

extern "C" void pushTarget(int streamId, size_t nSamples, size_t offset)
{
    if ( nSamples == 0 )
        nSamples = latest_target_size;
    if ( streamId < 0 )
        CHECK_CUDA_ERRORS(hipMemcpy(d_target+offset, target+offset, nSamples * sizeof(scalar), hipMemcpyHostToDevice))
    else
        CHECK_CUDA_ERRORS(hipMemcpyAsync(d_target+offset, target+offset, nSamples * sizeof(scalar), hipMemcpyHostToDevice, getLibStream(streamId)))
}

extern "C" void resizeOutput(size_t newSize)
{
    resizeArrayPair(timeseries, d_timeseries, timeseries_size, newSize);
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dd_timeseries), &d_timeseries, sizeof(scalar*)));
    latest_timeseries_size = newSize;
}

extern "C" void pullOutput(int streamId)
{
    if ( streamId < 0 )
        CHECK_CUDA_ERRORS(hipMemcpy(timeseries, d_timeseries, latest_timeseries_size * sizeof(scalar), hipMemcpyDeviceToHost))
    else
        CHECK_CUDA_ERRORS(hipMemcpyAsync(timeseries, d_timeseries, latest_timeseries_size * sizeof(scalar), hipMemcpyDeviceToHost, getLibStream(streamId)))
}

extern "C" void resizeSummary(size_t newSize)
{
    resizeArrayPair(summary, d_summary, summary_size, newSize);
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dd_summary), &d_summary, sizeof(scalar*)));
    latest_summary_size = newSize;
}

extern "C" void pullSummary(int streamId, size_t nSamples, size_t offset)
{
    if ( nSamples == 0 )
        nSamples = latest_summary_size;
    if ( streamId < 0 )
        CHECK_CUDA_ERRORS(hipMemcpy(summary+offset, d_summary+offset, nSamples * sizeof(scalar), hipMemcpyHostToDevice))
    else
        CHECK_CUDA_ERRORS(hipMemcpyAsync(summary+offset, d_summary+offset, nSamples * sizeof(scalar), hipMemcpyHostToDevice, getLibStream(streamId)))
}

extern "C" void libSync(unsigned int streamId)
{
    if ( streamId )
        CHECK_CUDA_ERRORS(hipStreamSynchronize(getLibStream(streamId)))
    else
        CHECK_CUDA_ERRORS(hipDeviceSynchronize())
}

extern "C" void libResetEvents(unsigned int nExpected)
{
    nextEvent = 0;
    getLibEvent(nExpected);
}

extern "C" unsigned int libRecordEvent(unsigned int streamId)
{
    hipEventRecord(getLibEvent(nextEvent), getLibStream(streamId));
    return nextEvent++;
}

extern "C" void libWaitEvent(unsigned int eventHandle, unsigned int streamId)
{
    hipStreamWaitEvent(getLibStream(streamId), getLibEvent(eventHandle), 0);
}

#endif
