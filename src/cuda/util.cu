#include "hip/hip_runtime.h"
#include "lib_definitions.h"

void pushDeltabar(std::vector<double> dbar)
{
    scalar h_deltabar[NPARAMS];
    for ( int i = 0; i < NPARAMS; i++ )
        h_deltabar[i] = dbar[i];
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(deltabar), h_deltabar, NPARAMS*sizeof(scalar)));
}

std::vector<unsigned short> pushDetuneIndices(int trajLen, int nTraj, const MetaModel &model)
{
    std::vector<int> detuneIndices = model.get_detune_indices(trajLen, nTraj);
    std::vector<unsigned char> h_detuneParamIndices(detuneIndices.size());
    std::vector<unsigned short> nDetunes(NPARAMS);
    for ( int i = 0; i < NPARAMS; i++ )
        nDetunes[i] = 0;
    for ( size_t i = 0; i < detuneIndices.size(); i++ ) {
        if ( detuneIndices[i] >= 0 )
            ++nDetunes[detuneIndices[i]];
        h_detuneParamIndices[i] = detuneIndices[i]; // Note, the negative indices are never consumed, so unsigned is not an error.
    }
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(detuneParamIndices), h_detuneParamIndices.data(), detuneIndices.size() * sizeof(unsigned char)));
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(numDetunesByParam), nDetunes.data(), NPARAMS * sizeof(unsigned short)));
    return nDetunes;
}



__global__ void observe_no_steps_kernel(int blankCycles)
{
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
    if ( id >= NMODELS )
        return;
    iStimulation stim = dd_stimUNI[id];
    iObservations obs = {};
    int tStart = 0;
    int nextObs = 0;
    if ( blankCycles > 0 ) {
        for ( const auto step : stim ) {
            if ( step.t > stim.duration )
                break;
            if ( !step.ramp ) {
                if ( tStart < step.t ) {
                    obs.start[nextObs] = tStart;
                    obs.stop[nextObs] = step.t;
                    if ( ++nextObs == iObservations::maxObs )
                        break;
                }
                tStart = step.t + blankCycles;
            }
        }
    }
    if ( nextObs < iObservations::maxObs ) {
        if ( tStart < stim.duration ) {
            obs.start[nextObs] = tStart;
            obs.stop[nextObs] = stim.duration;
        }
    }
    dd_obsUNI[id] = obs;
}

extern "C" void observe_no_steps(int blankCycles)
{
    dim3 block(256);
    observe_no_steps_kernel<<<((NMODELS+block.x-1)/block.x)*block.x, block.x>>>(blankCycles);
}



extern "C" void genRandom(unsigned int n, scalar mean, scalar sd, unsigned long long seed)
{
    resizeArray(d_random, random_size, n * sizeof(scalar));
    CHECK_CUDA_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dd_random), &d_random, sizeof(scalar*)));

    if ( seed != 0 )
        CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(cuRNG, seed));

#ifdef USEDOUBLE
    CURAND_CALL(hiprandGenerateNormalDouble(cuRNG, d_random, n, mean, sd));
#else
    CURAND_CALL(hiprandGenerateNormal(cuRNG, d_random, n, mean, sd));
#endif

}
